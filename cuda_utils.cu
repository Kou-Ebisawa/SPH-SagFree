#include "hip/hip_runtime.h"
/*! 
@file cuda_utils.cu

@brief CUDA���ʃf�o�C�X�֐�
	- CUDA�f�o�C�X(__device__)�֐����L�q����t�@�C��
	- cu�t�@�C������C���N���[�h����
	- �r���h����͏��O����

@author Makoto Fujisawa
@date 2023-02
*/

#ifndef _CUDA_UTILS_CU_
#define _CUDA_UTILS_CU_


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <stdio.h>
#include <math.h>

#include "hip/hip_vector_types.h"
#include <hip/hip_math_constants.h>

#include "cuda_utils.h"

//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <cstdio>
#include <GL/glew.h>
#if __APPLE__
	#include <OpenGL/gl.h>
	#include <OpenGL/glu.h>
#else
	#include <GL/gl.h>
	#include <GL/glu.h>
#endif

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>


//-----------------------------------------------------------------------------
// device�֐� - �f�o�C�X(GPU)�Ŏ��s�E�f�o�C�X�֐�����̂݌Ăяo����
//-----------------------------------------------------------------------------
/*!
* [a,b]�ɃN�����v
* @param[in] x �N�����v���������l
* @param[in] a,b �N�����v���E
* @return �N�����v���ꂽ���l
*/
__device__ 
inline float CxClamp(float x, float a, float b)
{
    return max(a, min(b, x));
}
__device__ 
inline int CxClamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

/*!
* a/b�̌v�Z���ʂ�؂�グ
* @param[in] a,b a/b
* @return �؂�グ�����Z����
*/
__device__
inline uint DivCeil(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

/*!
* �[������ for float3
* @param[in] v �l
*/
__device__
inline int CuIsZero(float3 v)
{
    if(fabsf(v.x) < 1.0e-10 && fabsf(v.y) < 1.0e-10 && fabsf(v.z) < 1.0e-10){
        return 1;
    }
    else{
        return 0;
    }
}

/*!
* �s��ƃx�N�g���̐�
* @param[in] m 3x3�s��(float3�̑傫��3�̔z��)
* @param[in] v 3D�x�N�g��
* @return �ς̌���
*/
__device__
inline float3 CuMulMV(float3 *m, float3 v)
{
    return make_float3(dot(m[0], v), dot(m[1], v), dot(m[2], v));
}



//-----------------------------------------------------------------------------
// �O���b�h
//-----------------------------------------------------------------------------
/*!
* 1D�C���f�b�N�X����3D�C���f�b�N�X�ւ̕ϊ�(�O���b�h���͔C��)
* @param[in] i 1D�C���f�b�N�X
* @param[in] gridSize �O���b�h��
* @return 3D�C���f�b�N�X
*/
__device__
inline int3 calcGridPos(int i, int3 ngrid)
{
	int3 gridPos;
	int w = i%(ngrid.x*ngrid.y);
	gridPos.x = w%ngrid.x;
	gridPos.y = w/ngrid.x;
	gridPos.z = i/(ngrid.x*ngrid.y);
	return gridPos;
}
/*!
* 3D�C���f�b�N�X����1D�C���f�b�N�X�ւ̕ϊ�(�O���b�h���͔C��)
* @param[in] p 3D�C���f�b�N�X
* @param[in] gridSize �O���b�h��
* @return 1D�C���f�b�N�X
*/
__device__
inline uint calcGridIndex(int3 p, int3 ngrid)
{
	p.x = min(p.x, ngrid.x-1);
	p.y = min(p.y, ngrid.y-1);
	p.z = min(p.z, ngrid.z-1);
	return (p.z*ngrid.x*ngrid.y)+(p.y*ngrid.x)+p.x;
}

//----------------------
// �Փ˔���
//----------------------
/*!
* AABB�Ɠ_�̋���
* @param[in] p �_���W
* @param[in] box_cen AABB�̒��S
* @param[in] box_ext AABB�̊e�ӂ̒�����1/2
* @param[out] cp AABB�\�ʂ̍ŋߖT�_
* @param[out] d ���s��AABB�̋���
* @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
*/
__device__
inline int distPointAABB(float3 p, float3 box_cen, float3 box_ext, float3& cp, float& d, float3& n)
{
    cp = p-box_cen;

    float3 tmp = fabs(cp)-box_ext;
    float res = ((tmp.x > tmp.y && tmp.x > tmp.z) ? tmp.x : (tmp.y > tmp.z ? tmp.y : tmp.z));

    float sgn = (res > 0.0) ? -1.0 : 1.0;

    int coli = 0;
    n = make_float3(0.0f);

    if(cp.x > box_ext.x){
        cp.x = box_ext.x;
        n.x -= 1.0;
        coli++;
    } else if(cp.x < -box_ext.x){
        cp.x = -box_ext.x;
        n.x += 1.0;
        coli++;
    }

    if(cp.y > box_ext.y){
        cp.y = box_ext.y;
        n.y -= 1.0;
        coli++;
    } else if(cp.y < -box_ext.y){
        cp.y = -box_ext.y;
        n.y += 1.0;
        coli++;
    }

    if(cp.z > box_ext.z){
        cp.z = box_ext.z;
        n.z -= 1.0;
        coli++;
    } else if(cp.z < -box_ext.z){
        cp.z = -box_ext.z;
        n.z += 1.0;
        coli++;
    }

    n = normalize(n);

    cp += box_cen;
    d = sgn*length(cp-p);

    return coli;
}

/*!
* �����Ɖ~�̌�������(2D, A��)
* @param[in] A,B �����̗��[�_���W
* @param[in] C �~�̒��S
* @param[in] r �~�̔��a
* @param[out] P ��_���W
* @return ��_��
*/
__device__ 
static int CuLineCircleIntersection(float2 A, float2 B, float2 C, float r, float2 P[2], float t[2])
{
	float rr = r*r;
	float2 AC = C-A;
	float2 BC = C-B;

	float2 v = B-A;
	float l = length(v);
	v /= l;

	float td = dot(v, AC);
	float2 D = A+td*v;
	float dd = dot(D-C, D-C);

	if(dd < rr){
		float dt = sqrtf(rr-dd);

		float da = rr-dot(AC, AC);
		float db = rr-dot(BC, BC);

		int inter = 0;
		float t1 = td-dt;
		float t2 = td+dt;
		if(t1 >= 0 && t1 <= l){
			P[inter] = A+t1*v;
			t[inter] = t1;
			inter++;
		}
		if(t2 >= 0 && t2 <= l){
			P[inter] = A+t2*v;
			t[inter] = t2;
			inter++;
		}

		return inter;
	}
	else{
		return 0;
	}
}


/*!
* AABB�Ƌ��̋���
* @param[in] spos �����S
* @param[in] r �����a
* @param[in] sgn
* @param[in] box_min,box_max AABB�ŏ��C�ő���W�l
* @param[out] cp AABB�\�ʂ̍ŋߖT�_
* @param[out] d ���s��AABB�̋���
* @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
*/
__device__
inline int collisionSphereAABB(float3 spos, float r, int sgn, float3 box_min, float3 box_max, float3 &cp, float &d, float3 &n)
{
	float3 dist_min;	// box_min�Ƃ̋���
	float3 dist_max;	// box_max�Ƃ̋���
	float d0 = 0.0f;
	float3 n0 = make_float3(0.0f, 0.0f, 0.0f);
	int bout = 0;
	int count = 0;

	// �e�����Ƃɍŏ��ƍő勫�E�O�ɂȂ��Ă��Ȃ������ׂ�
	if((dist_min.x = (spos.x-r)-box_min.x) < 0.0){ bout |= 0x0001; count++; d0 = dist_min.x; n0 = make_float3( 1.0,  0.0,  0.0);}
	if((dist_min.y = (spos.y-r)-box_min.y) < 0.0){ bout |= 0x0002; count++; d0 = dist_min.y; n0 = make_float3( 0.0,  1.0,  0.0);}
	if((dist_min.z = (spos.z-r)-box_min.z) < 0.0){ bout |= 0x0004; count++; d0 = dist_min.z; n0 = make_float3( 0.0,  0.0,  1.0);}
	if((dist_max.x = box_max.x-(spos.x+r)) < 0.0){ bout |= 0x0008; count++; d0 = dist_max.x; n0 = make_float3(-1.0,  0.0,  0.0);}
	if((dist_max.y = box_max.y-(spos.y+r)) < 0.0){ bout |= 0x0010; count++; d0 = dist_max.y; n0 = make_float3( 0.0, -1.0,  0.0);}
	if((dist_max.z = box_max.z-(spos.z+r)) < 0.0){ bout |= 0x0020; count++; d0 = dist_max.z; n0 = make_float3( 0.0,  0.0, -1.0);}

	// �����̓�(�S���ŋ��E��)
	if(bout == 0){
		float min_d = 1e10;
		if(dist_min.x < min_d){ min_d = dist_min.x; n = make_float3( 1.0,  0.0,  0.0); }
		if(dist_min.y < min_d){ min_d = dist_min.y; n = make_float3( 0.0,  1.0,  0.0); }
		if(dist_min.z < min_d){ min_d = dist_min.z; n = make_float3( 0.0,  0.0,  1.0); }

		if(dist_max.x < min_d){ min_d = dist_max.x; n = make_float3(-1.0,  0.0,  0.0); }
		if(dist_max.y < min_d){ min_d = dist_max.y; n = make_float3( 0.0, -1.0,  0.0); }
		if(dist_max.z < min_d){ min_d = dist_max.z; n = make_float3( 0.0,  0.0, -1.0); }

		d = (float)sgn*min_d;
		n *= (float)sgn;
		cp = spos+n*fabs(d);
		return 1;
	}

	// �����̊O
	// sgn = 1:���C-1:�I�u�W�F�N�g
	if(count == 1){
		// ���ʋߖT
		d = (float)sgn*d0;
		n = (float)sgn*n0;
		cp = spos+n*fabs(d);
	}
	else{
		// �G�b�W/�R�[�i�[�ߖT
		float3 x = make_float3(0.0f, 0.0f, 0.0f);
		if(bout & 0x0001) x.x =  dist_min.x;
		if(bout & 0x0002) x.y =  dist_min.y;
		if(bout & 0x0004) x.z =  dist_min.z;
		if(bout & 0x0008) x.x = -dist_max.x;
		if(bout & 0x0010) x.y = -dist_max.y;
		if(bout & 0x0020) x.z = -dist_max.z;

		d = length(x);
		n = normalize(x);

		d *= -(float)sgn;
		n *= -(float)sgn;

		cp = spos+n*fabs(d);

		float3 disp = make_float3(0.00001);
		//Random(disp, 0, 0.00001);
		disp = disp*n;
		cp += disp;
	}

	return 0;
}


/*!
* AABB�Ɠ_�̋���
* @param[in] p �_���W
* @param[in] box_cen AABB�̒��S
* @param[in] box_ext AABB�̊e�ӂ̒�����1/2
* @param[out] cp AABB�\�ʂ̍ŋߖT�_
* @param[out] d ���s��AABB�̋���
* @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
*/
__device__
inline int collisionPointAABB(float3 p, float3 box_cen, float3 box_ext, float3 &cp, float &d, float3 &n)
{
	cp = p-box_cen;

	float3 tmp = fabs(cp)-box_ext;
	float res = ((tmp.x > tmp.y && tmp.x > tmp.z) ? tmp.x : (tmp.y > tmp.z ? tmp.y : tmp.z));

	float sgn = (res > 0.0) ? -1.0 : 1.0;

	int coli = 0;
	n = make_float3(0.0f);

	if(cp.x > box_ext.x){
		cp.x = box_ext.x;
		n.x -= 1.0;
		coli++;
	}
	else if(cp.x < -box_ext.x){
		cp.x = -box_ext.x;
		n.x += 1.0;
		coli++;
	}

	if(cp.y > box_ext.y){
		cp.y = box_ext.y;
		n.y -= 1.0;
		coli++;
	}
	else if(cp.y < -box_ext.y){
		cp.y = -box_ext.y;
		n.y += 1.0;
		coli++;
	}

	if(cp.z > box_ext.z){
		cp.z = box_ext.z;
		n.z -= 1.0;
		coli++;
	}
	else if(cp.z < -box_ext.z){
		cp.z = -box_ext.z;
		n.z += 1.0;
		coli++;
	}

	n = normalize(n);

	//if(coli > 1){
	//	float3 disp;
	//	Random(disp, 0, 0.00001);
	//	disp = disp*n;
	//	cp += disp;
	//}

	cp += box_cen;
	d = sgn*length(cp-p);

	return 0;
}


/*!
* �_��BOX�̋���
* @param[in] p �_���W
* @param[in] box_cen BOX�̒��S
* @param[in] box_ext BOX�̊e�ӂ̒�����1/2
* @param[in] box_rot BOX�̕����s��(3x3��]�s��)
* @param[in] box_inv_rot BOX�̕����s��̋t�s��(3x3)
* @param[out] cp BOX�\�ʂ̍ŋߖT�_
* @param[out] d �_��BOX�̋���
* @param[out] n ��_�ɂ�����P�ʖ@���x�N�g��
*/
__device__
inline int collisionPointBox(float3 p, float3 box_cen, float3 box_ext, float3 box_rot[3], float3 box_inv_rot[3], float3 &cp, float &d, float3 &n)
{
	cp = p-box_cen;
	cp = CuMulMV(box_rot, cp);

	float3 tmp = fabs(cp)-box_ext;

	int coli = 0;
	n = make_float3(0.0f);

	if(tmp.x < 0.0 && tmp.y < 0.0 && tmp.z < 0.0){
		tmp = fabs(tmp);

		if(tmp.x <= tmp.y && tmp.x <= tmp.z){	// x���ʂɋ߂�
			if(cp.x > 0){
				cp.x = box_ext.x;
				n.x += 1.0;
			}
			else{
				cp.x = -box_ext.x;
				n.x -= 1.0;
			}
		}
		else if(tmp.y <= tmp.x && tmp.y <= tmp.z){ // y���ʂɋ߂�
			if(cp.y > 0){
				cp.y = box_ext.y;
				n.y += 1.0;
			}
			else{
				cp.y = -box_ext.y;
				n.y -= 1.0;
			}
		}
		else{ // z���ʂɋ߂�
			if(cp.z > 0){
				cp.z = box_ext.z;
				n.z += 1.0;
			}
			else{
				cp.z = -box_ext.z;
				n.z -= 1.0;
			}
		}

		coli++;
	}

	cp = CuMulMV(box_inv_rot, cp);
	n  = CuMulMV(box_inv_rot, n);

	n = normalize(n);
	cp += box_cen;

	float sgn = (coli) ? -1.0 : 1.0;
	d = sgn*(length(cp-p));

	return 0;
}

/*!
* �_�Ƌ��̋���
* @param[in] p �_���W
* @param[in] sphere_cen ���̒��S
* @param[in] sphere_rad ���̔��a
* @param[out] cp �_�Ƌ����S�����Ԑ����Ƌ��̌�_
* @param[out] d �_�Ƌ��\�ʂ̋���
* @param[out] n �����S����_�ւ̒P�ʃx�N�g��
*/
__device__
inline int collisionPointSphere(float3 p, float3 sphere_cen, float sphere_rad, float3 &cp, float &d, float3 &n)
{
	n = make_float3(0.0f);

	float3 l = p-sphere_cen;
	float ll = length(l);

	d = ll-sphere_rad;
	if(d < 0.0){
		n = normalize(p-sphere_cen);
		cp = sphere_cen+n*sphere_rad;
	}

	return 0;
}

/*!
* �_�ƕ��ʂ̋���
* @param[in] v  �_�̍��W
* @param[in] px ���ʏ�̓_
* @param[in] pn ���ʂ̖@��
* @return ����
*/
__device__ 
inline float distPointPlane(float3 v, float3 px, float3 pn)
{
	return dot((v-px), pn)/length(pn);
}

/*!
* �O�p�`�Ɠ_�̋����ƍŋߖT�_
* @param[in] v0,v1,v2	�O�p�`�̒��_
* @param[in] n			�O�p�`�̖@��
* @param[in] p			�_
* @return 
*/
__device__ 
inline int distPointTriangle(float3 v0, float3 v1, float3 v2, float3 n, float3 p, float &dist, float3 &p0)
{
	// �|���S�����܂ޕ��ʂƓ_�̋���
	float l = distPointPlane(p, v0, n);

	// ���ʂƂ̍ŋߖT�_���W
	float3 np = p-l*n;

	// �ߖT�_���O�p�`�����ǂ����̔���
	float3 n1 = cross((v0-p), (v1-p));
	float3 n2 = cross((v1-p), (v2-p));
	float3 n3 = cross((v2-p), (v0-p));

	if(dot(n1, n2) > 0 && dot(n2, n3) > 0){
		// �O�p�`��
		dist = l;
		p0 = np;
		return 1;
	}
	else{
		// �O�p�`�O
		return 0;
	}
}


/*!
* ���C/�����ƎO�p�`�̌���
* @param[in] P0,P1 ���C/�����̒[�_or���C��̓_
* @param[in] V0,V1,V2 �O�p�`�̒��_���W
* @param[out] I ��_���W
* @retval 1 ��_I�Ō��� 
* @retval 0 ��_�Ȃ�
* @retval 2 �O�p�`�̕��ʓ�
* @retval -1 �O�p�`��"degenerate"�ł���(�ʐς�0�C�܂�C�������_�ɂȂ��Ă���)
*/
inline __device__ 
int intersectSegmentTriangle(float3 P0, float3 P1, 
							 float3 V0, float3 V1, float3 V2, 
							 float3 &I, float3 &n, float rp = 0.01)
{
	// �O�p�`�̃G�b�W�x�N�g���Ɩ@��
	float3 u = V1-V0;		
	float3 v = V2-V0;			
	n = normalize(cross(u, v));
	if(CuIsZero(n)){
		return -1;	// �O�p�`��"degenerate"�ł���(�ʐς�0)
	}

	// ����
	float3 dir = P1-P0;
	float a = dot(n, P0-V0);
	float b = dot(n, dir);
	if(fabs(b) < 1e-10){	// �����ƎO�p�`���ʂ����s
		if(a == 0){
			return 2;	// ���������ʏ�
		}
		else{
			return 0;	// ��_�Ȃ�
		}
	}


	// ��_�v�Z

	// 2�[�_�����ꂼ��قȂ�ʂɂ��邩�ǂ����𔻒�
	float r = -a/b;
	if(a < 0){
		return 0;
	}

	if(r < 0.0){
		return 0;
	}
	else{
		if(fabs(a) > fabs(b)){
			return 0;
		}
		else{
			if(b > 0){
				return 0;
			}
		}
	}

	// �����ƕ��ʂ̌�_
	I = P0+r*dir;

	// ��_���O�p�`���ɂ��邩�ǂ����̔���
	float uu, uv, vv, wu, wv, D;
	uu = dot(u, u);
	uv = dot(u, v);
	vv = dot(v, v);
	float3 w = I-V0;
	wu = dot(w, u);
	wv = dot(w, v);
	D = uv*uv-uu*vv;

	float s, t;
	s = (uv*wv-vv*wu)/D;
	if(s < 0.0 || s > 1.0){
		return 0;
	}

	t = (uv*wu-uu*wv)/D;
	if(t < 0.0 || (s+t) > 1.0){
		return 0;
	}

	return 1;
}




/*!
* �����Ƌ��̌�������
* @param[in] s0,s1	�����̒[�_
* @param[in] sc,r   ���̒��S���W�Ɣ��a
* @param[out] d2 �����Ƃ̋����̓��
* @return ���������true
*/
__device__ 
inline bool segment_sphere(const float3 &s0, const float3 &s1, const float3 &sc, const float &r, float &d2)
{
	float3 v = s1-s0;
	float3 c = sc-s0;

	float vc = dot(v, c);
	if(vc < 0){		// ���̒��S�������̎n�_s0�̊O�ɂ���
		d2 = dot(c, c);
		return (d2 < r*r);	// �����S�Ǝn�_s0�̋����Ō�������
	}
	else{
		float v2 = dot(v, v);
		if(vc > v2){	// ���̒��S�������̏I�_s1�̊O�ɂ���
			d2 = dot(s1-sc, s1-sc);
			return (d2 < r*r);	// �����S�ƏI�_s1�̋����Ō�������
		}
		else{			// ����s0��s1�̊Ԃɂ���
			float3 a = (vc*v)/dot(v, v)-c;
			d2 = dot(a, a);
			return (d2 < r*r);	// �����Ƌ����S�̋����Ō�������
		}
	}
}

/*!
* ����(���܂ޒ���)�Ɠ_�̋���
* @param[in] v0,v1 �����̗��[�_���W
* @param[in] p �_�̍��W
* @return ����
*/
__device__ 
inline double segment_point_dist(const float3 &v0, const float3 &v1, const float3 &p)
{
	float3 v = normalize(v1-v0);
	float3 vp = p-v0;
	float3 vh = dot(vp, v)*v;
	return length(vp-vh);
}


/*!
* ����(���C,������)�Ƌ��̌�������
* @param[in] p,d ���C�̌��_�ƕ���
* @param[in] c,r ���̒��S�Ɣ��a
* @param[out] t1,t2 p�����_�܂ł̋���
* @return ��_��
*/
__device__
inline int ray_sphere(const float3 &p, const float3 &d, const float3 &sc, const float r, float &t1, float &t2)
{
	float3 q = p-sc;	// �����S���W�n�ł̌������_���W

	float a = dot(d, d);
	float b = 2*dot(q, d);
	float c = dot(q, q)-r*r;

	// ���ʎ�
	float D = b*b-4*a*c;

	if(D < 0.0){ // �����Ȃ�
		return 0;
	}
	else if(D < 1e-8){ // ��_��1
		t1 = -b/(2*a);
		t2 = -1;
		return 1;
	}
	else{ // ��_��2
		float sqrtD = sqrt(D);
		t1 = (-b-sqrtD)/(2*a);
		t2 = (-b+sqrtD)/(2*a);
		return 2;
	}

}
/*!
* �O�p�`�Ƌ��̌�������
* @param[in] v0,v1,v2	�O�p�`�̒��_
* @param[in] n			�O�p�`�̖@��
* @param[in] p			�ŋߖT�_
* @return 
*/
__device__
inline bool triangle_sphere(const float3 &v0, const float3 &v1, const float3 &v2, const float3 &n, 
							const float3 &c, const float &r, float &dist, float3 &ipoint)
{
	// �|���S�����܂ޕ��ʂƋ����S�̋���
	float d = dot(v0, n);
	float l = dot(n, c)-d;

	dist = l;
	if(l > r) return false;

	// ���ʂƂ̍ŋߖT�_���W
	float3 p = c-l*n;

	// �ߖT�_���O�p�`�����ǂ����̔���
	float3 n1 = cross((v0-c), (v1-c));
	float3 n2 = cross((v1-c), (v2-c));
	float3 n3 = cross((v2-c), (v0-c));

	ipoint = p;
	dist = l;
	if(dot(n1, n2) > 0 && dot(n2, n3) > 0){		// �O�p�`��
		return true;
	}
	else{		// �O�p�`�O
				// �O�p�`�̊e�G�b�W�Ƌ��̏Փ˔���
		for(int e = 0; e < 3; ++e){
			float3 va0 = (e == 0 ? v0 : (e == 1 ? v1 : v2));
			float3 va1 = (e == 0 ? v1 : (e == 1 ? v2 : v0));

			float t1, t2;
			int n = ray_sphere(va0, normalize(va1-va0), c, r, t1, t2);

			if(n){
				float le2 = dot(va1-va0, va1-va0);
				if((t1 >= 0.0 && t1*t1 < le2) || (t2 >= 0.0 && t2*t2 < le2)){
					return true;
				}
			}
		}
		return false;
	}
}


#endif // #ifndef _CUDA_UTILS_CU_