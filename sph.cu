#include "hip/hip_runtime.h"
/*! 
  @file sph.cu
	
  @brief CUDA : SPH�@

  @author Makoto Fujisawa
  @date 2023-02
*/


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include <cstdio>
#include <GL/glew.h>
#if __APPLE__
	#include <OpenGL/gl.h>
	#include <OpenGL/glu.h>
#else
	#include <GL/gl.h>
	#include <GL/glu.h>
#endif

#include "sph_kernel.cu"

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>

//-----------------------------------------------------------------------------
// CUDA�֐�
//-----------------------------------------------------------------------------
extern "C"
{
/*!
 * �p�����[�^��GPU�֓]��
 * @param[in] hparams �z�X�g(CPU)�������Ɋi�[���ꂽ�p�����[�^
 */
void CuSetParameters(const SceneParameter* hparams)
{
	CUCHECK(hipMemcpyToSymbol(HIP_SYMBOL(params), hparams, sizeof(SceneParameter), 0, hipMemcpyHostToDevice));
}

/*!
 * �X���b�h������u���b�N/�O���b�h���̌v�Z(�K�v�X���b�h��n�ȏ�ɂȂ�悤�ɐݒ�)
 * @param[in] n �K�v�X���b�h��
 * @param[out] block,grid �u���b�N��/�O���b�h��
 */
void CuCalGridN(int n, dim3& block, dim3& grid)
{
	// �X���b�h���̐ݒ�(n�ȏ�ɂȂ�悤�ɐݒ�)
	block = dim3(THREAD_NUM, 1, 1); // 1�u���b�N����X���b�h��
	grid = dim3((n+block.x-1)/block.x, 1, 1); // 1�O���b�h������̃u���b�N��
}

/*!
 * CUDA�J�[�l�����g���ĕ���v�Z:���q���x�̌v�Z
 *  - ���q�ʒu��cell.dSortedPos����擾����̂ň����Ƃ��ēn���K�v�Ȃ�
 * @param[out] ddens ���q���x(�f�o�C�X������)
 * @param[in] dvol ���q�̐�(�f�o�C�X������)
 * @param[in] n ���q��
 */
void CuSphDensity(float* drestdens,float* ddens, float* dvol,float* dmas, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphDensity<<<grid, block>>>(drestdens,ddens, dvol,dmas, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}

/*!
 * CUDA�J�[�l�����g���ĕ���v�Z:���q���͒l�𖧓x����v�Z
 * @param[out] dpres ���q����(�f�o�C�X������)
 * @param[in] ddens ���q���x(�f�o�C�X������)
 * @param[in] n ���q��
 */
void CuSphPressure(float* drestdens,float* dpres, float* ddens, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphPressure<<<grid, block>>>(drestdens,dpres, ddens, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}

/*!
* CUDA�J�[�l�����g���ĕ���v�Z:�e���q�̉Q�x�v�Z[Macklin2013]
*  - M. Macklin & M. M\"{u}ller, "Position Based Fluids", ACM ToG, 32(4), pp.104:1-104:12, 2013.
*  - �O���b�h�@�����̌��̎�@�� 
*    R. Fedkiw; J. Stam & H. Jensen, "Visual simulation of smoke", Proc. SIGGRAPH 2001, pp.15-22, 2001.
* @param[out] dvort �e���q�̉Q�x�x�N�g��(�f�o�C�X������)
* @param[in] dvel ���q���x�z��(�f�o�C�X������)
* @param[in] ddens ���q���x(�f�o�C�X������)
* @param[in] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ���,1�ŋ��E)(�f�o�C�X������)
* @param[in] n ���q��
*/
void CuSphVorticity(float* dvort, float* dvel, float* ddens, float* dvol, int* datt, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphVorticity<<<grid, block>>>(dvort,dvel, ddens, dvol, datt, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}

/*!
 * CUDA�J�[�l�����g���ĕ���v�Z:���q�ɓ�����(���͍�&�O�͍�)�̌v�Z
 * @param[out] dacc �e���q�ɓ�����(�����xdv/dt)(�f�o�C�X������)
 * @param[in] dvel ���q���x�z��(�f�o�C�X������)
 * @param[in] ddens ���q���x(�f�o�C�X������)
 * @param[in] dpres ���q����(�f�o�C�X������)
 * @param[in] dvort �e���q�̉Q�x�x�N�g��(�f�o�C�X������)
 * @param[in] dvol  ���q�̐�(�f�o�C�X������)
 * @param[in] datt ���q����(0�ŗ���,1�ŋ��E)(�f�o�C�X������)
 * @param[in] n ���q��
 */
void CuSphForces(float* drestdens,float* dacc, float* dvel, float* ddens, float* dpres, float* dvort, float* dvol,float* dmas, int* datt,float3 power,float* dfss, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphForces<<<grid, block>>>(drestdens,dacc, dvel, ddens, dpres, dvort, dvol,dmas, datt,power,dfss, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}
/*!
* CUDA�J�[�l�����g���ĕ���v�Z:���q�ɓ�����(�S����)�̌v�Z[Becker2007]
*  - M. Becker & M. Teschner, "Weakly Compressible SPH for Free Surface Flows", Proc. SCA2007, pp.209-217, 2007.
* @param[inout] dacc �e���q�ɓ�����(�����xdv/dt)(�f�o�C�X������)
* @param[in] dvel ���q���x�z��(�f�o�C�X������)
* @param[in] ddens ���q���x(�f�o�C�X������)
* @param[in] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)(�f�o�C�X������)
* @param[in] n ���q��
*/
void CuSphViscosityForces(float* drestdens,float* dacc, float* dvel, float* ddens, float* dvol,float* dmas, int* datt, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphViscosity<<<grid, block>>>(drestdens,dacc, dvel, ddens, dvol,dmas, datt, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}

/*!
* CUDA�J�[�l�����g���ĕ���v�Z:XSPH�l�H�S���̌v�Z[Schechter2012]
*  - H. Schechter & R. Bridson, "Ghost SPH for animating water", ACM ToG, 31(4), pp.61:1-61:8, 2012.
*  - ��(�����x)�Ƃ��ĔS������������̂ł͂Ȃ��C���x�𒼐ڍX�V����
*  - ���̂̕����I�����Ƃ��Ă̔S���ł͂Ȃ��v�Z���萫�̂��߂̌v�Z�Ƃ����������悳����
* @param[inout] dvel ���q���x�z��(�f�o�C�X������)
* @param[in] ddens ���q���x(�f�o�C�X������)
* @param[in] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)(�f�o�C�X������)
* @param[in] n ���q��
*/
void CuSphXSPHViscosity(float* dvel, float* ddens, float* dvol,float* dmas, int* datt, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphXSPHViscosity<<<grid, block>>>(dvel, ddens, dvol,dmas, datt, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}


/*!
 * CUDA�J�[�l�����g���ĕ���v�Z:�����x�ɏ]���Ĉʒu�Ƒ��x���X�V
 * @param[inout] dpos ���q�ʒu�z��(�f�o�C�X������)
 * @param[inout] dvel ���q���x�z��(�f�o�C�X������)
 * @param[in] dacc �e���q�ɓ�����(�����x)���i�[�����z��(�f�o�C�X������)
 * @param[in] dvol ���q�̐�(�f�o�C�X������)
 * @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)(�f�o�C�X������)
 * fix:�C�V��ǉ� �Œ�_��\��
 * @param[in] n ���q��
 */
void CuSphIntegrate(float* dpos, float* dvel, float* dacc, int* datt, int* dfix,int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphIntegrate<<<grid, block>>>(dpos, dvel, dacc, datt,dfix, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}


/*!
* CUDA�J�[�l�����g���ĕ���v�Z:�����x�ɏ]���đ��x�݂̂��X�V
*  - XSPH�p
* @param[inout] dvel ���q���x�z��(�f�o�C�X������)
* @param[in] dacc �e���q�ɓ�����(�����x)���i�[�����z��(�f�o�C�X������)
* @param[in] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)(�f�o�C�X������)
* @param[in] n ���q��
*/
void CuSphIntegrateV(float* dvel, float* dacc, int* datt, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphIntegrateVelocity<<<grid, block>>>(dvel, dacc, datt, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}
/*!
* CUDA�J�[�l�����g���ĕ���v�Z:���x���]���Ĉʒu���X�V
*  - XSPH�p
* @param[inout] dpos ���q�ʒu�z��(�f�o�C�X������)
* @param[in] dvel ���q���x�z��(�f�o�C�X������)
* @param[in] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)(�f�o�C�X������)
* @param[in] n ���q��
*/
void CuSphIntegrateP(float* dpos, float* dvel, int* datt,int*dfix, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphIntegratePosition<<<grid, block>>>(dpos, dvel, datt,dfix, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}



/*!
* CUDA�J�[�l�����g���ĕ���v�Z:���q�̐ς̌v�Z
*  - ���q�ʒu��cell.dSortedPos����擾����̂ň����Ƃ��ēn���K�v�Ȃ�
* @param[out] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)(�f�o�C�X������)
* @param[in] n �������闱�q��(offset����̑��ΓI�Ȉʒu)
* @param[in] v ���̗��q�̏ꍇ�̗��q�̐ϒl
*/
void CuSphCalVolume(float* dvol, int *datt, int n, float v)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxSphCalVolume<<<grid, block>>>(dvol, datt, n, v);	// �J�[�l�����s
	hipDeviceSynchronize();
}

/*!
* CUDA�J�[�l�����g���ĕ���v�Z:���q���x�̌v�Z
*  - ���q�ʒu��cell.dSortedPos����擾����̂ň����Ƃ��ēn���K�v�Ȃ�
* @param[out] dF ���x�l���i�[����O���b�h�Z���z��(�f�o�C�X������)
* @param[in] dvol ���q�̐�(�f�o�C�X������)
* @param[in] datt ���q����(0�ŗ���,1�ŋ��E)(�f�o�C�X������)
* @param[in] n ���q��
* @param[in] gnum �O���b�h��
* @param[in] gmin �O���b�h�ŏ����W
* @param[in] glen �O���b�h��
*/
void CuSphDensityInGrid(float* dF, float* dvol, int* datt, int n, int3 gnum, float3 gmin, float3 glen)
{
	// ���O���b�h�Z����=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	int numcell = gnum.x*gnum.y*gnum.z;
	dim3 block, grid;
	CuCalGridN(numcell, block, grid);	
	CxSphDensityAtCell<<<grid, block>>>(dF, dvol, datt, n, gnum, gmin, glen);	// �J�[�l�����s
	hipDeviceSynchronize();
}


/*!
* CUDA�J�[�l�����g���ĕ���v�Z:���q�̕`��F�𖧓x����v�Z
* @param[out] dcol ���q�F�z��(�f�o�C�X������)
* @param[in] dval  ���q�����ʔz��(�f�o�C�X������)
* @param[in] n ���q��
* @param[in] c1,c2 �����ʂ��ŏ�,�ő�̂Ƃ��̐F(�Ԃ̐F�͐��`��Ԃŋ��߂���)
* @param[in] range x�v�f�ɍŏ��l�Cy�v�f�ɍő�l
*/
void CuColorScalar(float* dcol, int* datt, float* dval, int n, float3 c1, float3 c2, float2 range)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxColorScalar<<<grid, block>>>(dcol, datt, dval, n, c1, c2, range);	// �J�[�l�����s
	hipDeviceSynchronize();
}
void CuColorVector(float* dcol, int* datt, float* dval, int n, float3 c1, float3 c2, float2 range)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxColorVector<<<grid, block>>>(dcol, datt, dval, n, c1, c2, range);	// �J�[�l�����s
	hipDeviceSynchronize();
}
/*!
* CUDA�J�[�l�����g���ĕ���v�Z:���q�̕`��F�ݒ� - ���̐F
* @param[out] dcol ���q�F�z��(�f�o�C�X������)
* @param[in] col �`��F
* @param[in] n ���q��
*/
void CuColorConstant(float* dcol, int* datt, float3 col, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxColorConstant<<<grid, block>>>(dcol, datt, col, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}


/*!
 * �e���q�̃O���b�h�n�b�V���l�v�Z(�ߖT�T���p)
 * @param[out] dhash �e���q�̃O���b�h�n�b�V���l���i�[�����z��
 * @param[out] dsortedidx �e���q�̃C���f�b�N�X���i�[�����z��(�ォ��n�b�V���l�Ń\�[�g����� -> �����_�ł͂܂��\�[�h�ς݂ł͂Ȃ�)
 * @param[in] dpos ���q�ʒu���i�[�����z��
 * @param[in] n ���q��
 */
void CuCalcHash(uint* dhash, uint* dindex, float* dpos, int n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxCalcHash<<<grid, block>>>(dhash, dindex, dpos, n);	// �J�[�l�����s
	hipDeviceSynchronize();
}

/*!
 * thrust::sort_by_key�ɂ��n�b�V���l�Ɋ�Â��\�[�g
 * @param[in] dhash �n�b�V���l
 * @param[in] dindex �C���f�b�N�X(�p�[�e�B�N���C�|���S���Ȃ�)
 * @param[in] n �f�[�^��
 */
void CuSort(unsigned int* dhash, uint* dindex, uint n)
{
	thrust::sort_by_key(thrust::device_ptr<unsigned int>(dhash),
					    thrust::device_ptr<unsigned int>(dhash+n),
					    thrust::device_ptr<unsigned int>(dindex));
	hipDeviceSynchronize();
}

/*!
 * �p�[�e�B�N���z����\�[�g���ꂽ���Ԃɕ��ёւ��C�e�Z���̎n�܂�ƏI���̃C���f�b�N�X������
 * @param[in] cell �ߖT�T���p�O���b�h�f�[�^
 * @param[in] dpos ���q�ʒu
 * @param[in] dvel ���q���x
 */
void CuReorderDataAndFindCellStart(Cell cell, float* dpos, float* dvel, uint n)
{
	dim3 block, grid;
	CuCalGridN(n, block, grid);	// ���q��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z

	// �Z���X�^�[�g�ʒu�z��̏�����
	CUCHECK(hipMemset(cell.dCellStart, 0xffffffff, cell.uNumCells*sizeof(uint)));

	// �V�F�A�[�h�������̃T�C�Y
	uint smemSize = sizeof(uint)*(THREAD_NUM+1);

	// �J�[�l�����s
	CxReorderDataAndFindCellStartD<<<grid, block, smemSize>>>(cell, dpos, dvel, n);
	hipDeviceSynchronize();
}

//�C�V��ǉ�-----------------------------------------------------------------------------------------------------
//XPBD�̐L�сE����f�C�Ȃ��E�˂��ꐧ��̏���
//dpos:�ʒu
//dmas:����
//dlen:���
//dkss:�L�э���
//dkbt:�Ȃ�����
//dquat:�p��(�l����)
//domega:��_���{�[�x�N�g��
//dlamb_ss:XPBD�̐L�сE����f����ɗp�����
//dlamb_bt:XPBD�̋Ȃ��E�˂��ꐧ��ɗp�����
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//dt:�^�C���X�e�b�v
//n:���q��
//iter:������
//example_flag:�`��ɂ���āC�������ꕔ�ς���
void CuXPBDConstraint(float* dpos,float* dcurpos,float* dmas, float* dlen, float* dkss,float* dkbt, float* dquat,float* dcurquat, float* domega, float* dlamb_ss,float* dlamb_bt,int* dfix, float dt,int n,int iter,bool example_flag) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	//XPBD�̏����̂��߂ɁC�ɂ�0�ɂ���
	CxSetLambdaZero << <grid, block >> > (dlamb_ss,dlamb_bt, n);
	hipDeviceSynchronize();
	//�L�ѐ���̔���
	for (int i = 0; i < iter; i++) {
		//�S�Ă̐���𓯎��Ɏ��s����ƁC�Փ˂��������邽�߁C��Ƌ����ɕ����Ď��s����
		//�����Ԗڂ�id�����s
		CxStretchingShearConstraint << <grid, block >> > (dpos, dcurpos, dmas, dlen, dkss, dquat, dcurquat, dlamb_ss, dfix, dt, n, 0, i, example_flag);
		//��Ԗڂ�id�����s
		CxStretchingShearConstraint << <grid, block >> > (dpos, dcurpos, dmas, dlen, dkss, dquat, dcurquat, dlamb_ss, dfix, dt, n, 1, i, example_flag);
		
		CxBendTwistConstraint << <grid, block >> > (dmas, dquat, dcurquat, domega, dkbt, dlamb_bt, dlen, dfix, dt, n, 0, i, example_flag);
		CxBendTwistConstraint << <grid, block >> > (dmas, dquat, dcurquat, domega, dkbt, dlamb_bt, dlen, dfix, dt, n, 1, i, example_flag);
	}
}

//�Փː���
//�����ł́C��Ԏ������e�Ղȋ��Ƃ̏Փ˂���������
//dpos:�ʒu
//dvel:���x
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//center:�є��Ƃ̏Փ˂������������̒��S
//rad:�є��Ƃ̏Փ˂������������̔��a
//dt:�^�C���X�e�b�v
//n:���q��
void CuCollisionConstraint(float* dpos, float* dvel, int* dfix, float3 center, float rad, float dt, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxCollisionConstraint << <grid, block >> > (dpos, dvel, dfix, center, rad, dt, n);
	hipDeviceSynchronize();
}

//�C�V��ǉ�
//���Ԑϕ�
//�ʒu�x�[�X�@�ɏ]���C���݂̈ʒu�ƈʒu�C����̈ʒu���瑬�x�����߁C�ʒu���X�V
//dpos:�ʒu(�ʒu�C����)
//dcurpos:�O�X�e�b�v�̈ʒu(�ʒu�C���O)
//dvel:���x
//dt:�^�C���X�e�b�v
//n:���q��
//vel_control:���ȉ��̑��x�̏ꍇ�ɐ؂�̂Ă��s�����ǂ������w��
void CuIntegrate(float* dpos,float* dcurpos,float* dvel,float dt,int n,bool vel_control) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxIntegrate << <grid, block >> > (dpos, dcurpos, dvel, dt, n, vel_control);
	hipDeviceSynchronize();
}

//�C�V��ǉ�
//����d�͂��C���[�W�����O�͌v�Z
//�f�o�b�N�p
void CuCalExternalForces(float* dpos,float*dvel,float* dmass,int* dfix,float3 gravity, float3 wind, float dt, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxCalExternalForces << <grid, block >> > (dpos, dvel, dmass, dfix, gravity, wind, dt, n);
	hipDeviceSynchronize();
}

//�C�V��ǉ�
//�ʒu�x�[�X�@(�g���ʒu�x�[�X�@�łȂ��C�L�сE����f����̂�)
//�f�o�b�N�p
void CuPBDStretchingConstraint(float* dpos, float* dmas, float* dlen, float* dkss, float* dquat, int* dfix, int n, int iter) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	for (int i = 0; i < iter; i++) {
		CxStretchingConstraint<<<grid,block>>>(dpos, dmas, dlen, dkss, dquat, dfix, n, 0);
		hipDeviceSynchronize();
		CxStretchingConstraint<<<grid,block>>>(dpos, dmas, dlen, dkss, dquat, dfix, n, 1);
	}
}

//�f�o�b�N�p�̔z�����ʒu�̏o��
void CuPrint3Dfloat(float* dpos,float* dvel,float* dacc,int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxPrint3Dfloat << <grid, block >> > (dpos, dvel, dacc, n);
	hipDeviceSynchronize();
}

//�ڐ��̍X�V
//kajiya-kay���f���ł̃����_�����O�ɗ��p
//dpos:�ʒu
//dtang:�G�b�W���Ƃ̐ڐ�
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��
void CuTangUpdate(float* dpos, float* dtang, int* dfix, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxTangUpdate << <grid, block >> > (dpos, dtang, dfix, n);
	hipDeviceSynchronize();
}

//�p���x�ȂǏ�������f�o�C�X�������ɐݒ�����Ă�����̂̏����l��0�ɐݒ�
//dangvel:�p���x
//dfss:�G�b�W���Ƃɂ������(GlobalForceStep�ŋ��߂�)
//dpbf_lambda:���x����̌v�Z�ߒ��ɕK�v�ȃɂ��������m��
//n:���q��
void CuSetParametersZero(float* dangvel, float* dfss, float* dpbf_lambda, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxSetParametersZero << <grid, block >> > (dangvel,dfss,dpbf_lambda, n);
	hipDeviceSynchronize();
}

//�p�����x�̍X�V
//dangvel:�p���x
//dquat:�p��(�l����)
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//dt:�^�C���X�e�b�v
//n:���q��
void CuAngVelUpdate(float* dangvel, float* dquat,int* dfix,float dt, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxAngVelUpdate << <grid, block >> > (dangvel, dquat,dfix, dt, n);
	hipDeviceSynchronize();
}

//�e�����x�̎��Ԑϕ�
//dangvel:�p���x
//dcurquat:�O�X�e�b�v�̎p��(�ʒu�C���O)
//dquat:���݂̎p��(�ʒu�C����)
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//dt:�^�C���X�e�b�v
//n:���q��
//vel_control:�p���x�����ȉ��Ȃ�؂�̂Ă��s�����ǂ����𔻒�
void CuAngVelIntegrate(float* dangvel,float* dcurquat, float* dquat,int* dfix,float dt, int n,bool vel_control) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxAngVelIntegrate << <grid, block >> > (dangvel, dcurquat, dquat, dfix, dt, n, vel_control);
	hipDeviceSynchronize();
}

//��ƂȂ閧�x�̐ݒ�
//dpos:�ʒu
//dRestDens:���q���Ƃɐݒ肷�����x
//dvol:�̐�
//n:���q��
void CuRestDensSet(float* dpos,float* dRestDens, float* dvol,float* dmas, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxRestDensSet << <grid, block >> > (dpos, dRestDens, dvol, dmas, n);
	hipDeviceSynchronize();
}

//�ꗥ�̊�ƂȂ閧�x�̐ݒ�
//�f�o�b�N�p
void CuRestTotalDens(float* drestdens,float dens, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxRestTotalDens << <grid, block >> > (drestdens, dens, n);
	hipDeviceSynchronize();
}

//�O���[�o���t�H�[�X�X�e�b�v
//�d�͂Ȃǂɂ��G�b�W�ɂ�����͂����߂�
//dfss:�G�b�W���Ƃɂ������
//dmass:����
//last_index:�є����Ƃ̍Ō�̗��q�̃C���f�b�N�X���i�[
//gravity:�d��
//num_elastic:�����ł́C�є����Ƃɕ���v�Z���邽�߁C�є��̐���n��
void CuGlobalForceStep(float* dpos,float* dfss,float* dmass, int* last_index, float3 gravity,float* ddens,float* drestdens,float* dvol, int num_elastic) {
	dim3 block, grid;
	CuCalGridN(num_elastic, block, grid);
	CxGlobalForceStep << <grid, block >> > (dpos,dfss, dmass, last_index, gravity, ddens, drestdens, dvol, num_elastic);
	hipDeviceSynchronize();
}

//���[�J���t�H�[�X�X�e�b�v
//�O���[�o���t�H�[�X�X�e�b�v�ŋ��߂��G�b�W���Ƃ̗͂���C�ό`��h�����߂̊����p�������߂�
//dpos:�ʒu
//dlen:���
//dquat:�p��
//dcurquat:�O�X�e�b�v�̎p��(�V�~�����[�V�����J�n�O��curquat��quat�ƈ�v���邽�߁C�X�V��̒l����)
//dkss:�L�э���
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��(�G�b�W���Ƃɕ���v�Z)
void CuLocalForceStep(float* dpos, float* dlen, float* dquat,float* dcurquat, float* dkss, float* dfss, int* dfix, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxLocalForceStep << <grid, block >> > (dpos, dlen, dquat, dcurquat, dkss, dfss, dfix, n);
	hipDeviceSynchronize();
}

//�O���[�o���g���N�X�e�b�v
//�є����ƂɃt�H�[�X�X�e�b�v�Ő������g���N��ł�������_���{�[�x�N�g�������߂�
//dpos:�ʒu
//dquat:�p��
//domega:��_���{�[�x�N�g��
//dlen:���
//dkss:�L�э���
//dkbt:�Ȃ�����
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//last_index:�є����Ƃ̍Ō�̗��q�̃C���f�b�N�X���i�[
//num_elastic:�����ł́C�є����Ƃɕ���v�Z���邽�߁C�є��̐���n��
void CuGlobalTorqueStep(float* dpos, float* dquat, float* domega, float* dlen, float* dkss, float* dkbt, int* dfix, int* dlast_index, int num_elastic) {
	dim3 block, grid;
	CuCalGridN(num_elastic, block, grid);
	//�����狁�߂�
	CxGlobalTorqueStep << <grid, block >> > (dpos, dquat, domega, dlen, dkss, dkbt, dfix, dlast_index, num_elastic);
	//�ォ�狁�߂�
	//CxGlobalTorqueStep_Upstair << <grid, block >> > (dpos, dquat, domega, dlen, dkss, dkbt, dfix, dlast_index, num_elastic);
	hipDeviceSynchronize();
}

//���[�J���g���N�X�e�b�v
//��_���{�[�x�N�g����K�؂Ȍ`�Ő��K������
//dquat:�p��
//domega:��_���{�[�x�N�g��
//deln:���
//dkbt:�Ȃ�����
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��(��_���{�[�x�N�g�����Ƃɕ���v�Z)
void CuLocalTorqueStep(float* dquat,float* domega, float* dlen, float* dkbt, int* dfix, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxLocalTorqueStep << <grid, block >> > (dquat, domega, dlen, dkbt, dfix, n);
	hipDeviceSynchronize();
}

//���x����̌v�Z
//pos:�ʒu
//ddens:���݂̖��x
//drestdens:����x
//dpbf_lambda:����ɗp�����
//dvol:�̐�
//n:���q��
void CuPbfConstraint(float* dpos,float* ddens,float* drestdens,float*dpbf_lambda,float*dvol,float* dmas,int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxSphDensity << <grid, block >> > (drestdens, ddens, dvol,dmas, n);//���x�v�Z
	CxPbfLambda << <grid, block >> > (ddens,drestdens, dpbf_lambda, dvol,dmas, n);//����ɗp����ɂ����߂�
	hipDeviceSynchronize();
	CxPbfConstraint << <grid, block >> > (dpos, drestdens, dpbf_lambda, dvol,dmas, n);//���񏈗�
	hipDeviceSynchronize();
}

//PBF�ŉ����ꍇ�̊O�͍��̌v�Z
//dacc:�����x
//datt:���q����(0�ŗ���,1�ŋ��E)
//power:���Ȃǂ̗�
//n:���q��
void CuPbfExternalForces(float* dacc, int* datt, float3 power,bool wind_flag, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxPbfExternalForces << <grid, block >> > (dacc, datt, power, wind_flag, n);
	hipDeviceSynchronize();
}

//���C����
void CuFrictionConstraint(float* dpos, float* dcurpos, float* drestdens, float* dvol, float* ddens, int* dfix, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	//�e���q�Ƃ̖��C�͂����v������C�Î~���C���ǂ����𔻒�
	//CxFrictionConstraint << <grid, block >> > (dpos, dcurpos, drestdens, dvol, ddens, dfix, n);
	//�e���q�ƐÎ~���C���𔻒肵����C���v
	CxFrictionAllParticlesConstraint << <grid, block >> > (dpos, dcurpos, drestdens, dvol, ddens, dfix, n);
	hipDeviceSynchronize();
}

//���C����̌�C�p�����C������
void CuFrictionConstraint_withQuat(float* dpos, float* dcurpos, float* drestdens, float* dvol, float* ddens, float* dquat, float* dlen, int* dfix, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxFrictionConstraint_withQuat << <grid, block >> > (dpos, dcurpos, drestdens, dvol, ddens, dquat, dlen, dfix, n);
	hipDeviceSynchronize();
}

//2���_����p����ݒ�
void CuQuatSet(float* dpos, float* dquat, int* dfix, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxQuatSet << <grid, block >> > (dpos, dquat, dfix, n);
	hipDeviceSynchronize();
}

//�g���N���v�Z������
void CuCalcTorque(float* dpos,float* dmas, float* dquat, float* dfss, float* dlength,float* dkss, int* dfix, float3 gravity, int n) {
	dim3 block, grid;
	CuCalGridN(n, block, grid);
	CxCalcTorque << <grid, block >> > (dpos, dmas, dquat, dfss, dlength, dkss, dfix, gravity, n);
	hipDeviceSynchronize();
}

//--------------------------------------------------------------------------------------------------------------------------

//-----------------------------------------------------------------------------
// GPU�����⏕�֐�
//-----------------------------------------------------------------------------
/*!
 * CUDA�f�o�C�X�̐ݒ� - id�𒼐ڎw��
 * @param[in] id �f�o�C�XID
 */
void CuSetDevice(int id)
{
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	if(id < 0 || id >= device_count){
		id = 0;
	}
	hipSetDevice(id);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, id);

	std::cout << " ---- GPU Info ----" << std::endl;
	std::cout << " Device        : " << prop.name << std::endl;
	std::cout << " Global mem    : " << prop.totalGlobalMem << " Byte" << std::endl;
	std::cout << " Constant mem  : " << prop.totalConstMem  << " Byte" << std::endl;
	std::cout << " Thresds/Block : " << prop.maxThreadsPerBlock << std::endl;
	std::cout << std::endl;
	THREAD_NUM = prop.maxThreadsPerBlock; // 1�u���b�N������̃X���b�h���ő�l
}

/*!
 * CUDA�f�o�C�X�̐ݒ�
 *  - �R�}���h���C�������Ɋ�Â�CUDA�f�o�C�X��ݒ�((��)-device 0)
 * @param[in] argc �R�}���h���C�������̐�
 * @param[in] argv �R�}���h���C���������X�g(argv[0]�͎��s�t�@�C����)
 */
void CuInit()
{
	CuSetDevice(0);
}


/*!
 * �f�o�C�X�������̊m��
 * @param[out] dPtr �f�o�C�X�������ւ̃|�C���^
 * @param[in] size �m�ۃT�C�Y(��������̃T�C�Y)
 */
void CuAllocateArray(void** dPtr, size_t size)
{
	CUCHECK(hipMalloc(dPtr, size));
}

/*!
 * �f�o�C�X�������̉��
 * @param[in] devPtr �f�o�C�X�������ւ̃|�C���^
 */
void CuFreeArray(void* dPtr)
{
	CUCHECK(hipFree(dPtr));
}

/*!
 * �f�o�C�X�������̈�̏�����
 * @param[in] dPtr �f�o�C�X�������ւ̃|�C���^
 * @param[in] val �����l
 * @param[in] size ����������̈�̃T�C�Y(��������̃T�C�Y)
 */
void CuSetArrayValue(void* dPtr, int val, size_t size)
{
	CUCHECK(hipMemset(dPtr, val, size));
}

/*!
 * �f�o�C�X�������ԃR�s�[
 * @param[in] dDst �R�s�[��
 * @param[in] dSrc �R�s�[��
 * @param[in] size �R�s�[�T�C�Y(��������̃T�C�Y)
 */
void CuCopyArrayD2D(void* dDst, void* dSrc, int size)
{
	CUCHECK(hipMemcpy(dDst, dSrc, size, hipMemcpyDeviceToDevice));
}


/*!
 * VBO���}�b�s���O
 * @param[in] vbo VBO,PBO��
 */
void* CuMapGLBufferObject(hipGraphicsResource** resource)
{
	void* ptr;
	CUCHECK(hipGraphicsMapResources(1, resource, 0));
	size_t num_bytes;
	CUCHECK(hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes, *resource));
	return ptr;
}

/*!
 * VBO���A���}�b�v
 * @param[in] vbo VBO,PBO��
 */
void CuUnmapGLBufferObject(hipGraphicsResource* resource)
{
	CUCHECK(hipGraphicsUnmapResources(1, &resource, 0));
}

/*!
 * PBO,VBO�o�b�t�@��CUDA�ɓo�^
 * @param[in] vbo VBO,PBO��
 */
void CuRegisterGLBufferObject(unsigned int vbo, hipGraphicsResource** resource)
{
	CUCHECK(hipGraphicsGLRegisterBuffer(resource, vbo, cudaGraphicsMapFlagsNone));
}

/*!
 * PBO,VBO�o�b�t�@��CUDA����폜
 * @param[in] vbo VBO,PBO��
 */
void CuUnregisterGLBufferObject(hipGraphicsResource* resource)
{
	CUCHECK(hipGraphicsUnregisterResource(resource));
}

/*!
 * �f�o�C�X����z�X�g�������ւ̃R�s�[
 * @param[in] hDst �R�s�[��z�X�g������(�Œ�size���m�ۂ���Ă��邱��)
 * @param[in] dSrc �R�s�[���f�o�C�X������
 * @param[in] vbo dSrc��VBO�̏ꍇ�CVBO��ID�D�����łȂ��ꍇ��0���w��
 * @param[in] size �R�s�[�T�C�Y(��������̃T�C�Y)
 */
void CuCopyArrayFromDevice(void* hDst, void* dSrc, hipGraphicsResource** resource, int offset, int size)
{
	if(resource) dSrc = CuMapGLBufferObject(resource);

	CUCHECK(hipMemcpy(hDst, (char*)dSrc+offset, size, hipMemcpyDeviceToHost));

	if(resource) CuUnmapGLBufferObject(*resource);
}

/*!
 * �z�X�g����f�o�C�X�������ւ̃R�s�[
 * @param[in] dDst �R�s�[��f�o�C�X������(�Œ�size���m�ۂ���Ă��邱��)
 * @param[in] hSrc �R�s�[���z�X�g������
 * @param[in] offset �R�s�[��I�t�Z�b�g
 * @param[in] size �R�s�[�T�C�Y(��������̃T�C�Y)
 */
void CuCopyArrayToDevice(void* dDst, const void* hSrc, int offset, int size)
{
	CUCHECK(hipMemcpy((char*)dDst+offset, hSrc, size, hipMemcpyHostToDevice));
}

/*!
 * �X���b�h����
 */
void CuThreadSync(void)
{
	CUCHECK(hipDeviceSynchronize());
}

/*!
 * �f�o�C�X�v���p�e�B�̕\��
 */
void CuDeviceProp(void)
{
	int n;	//�f�o�C�X��
	CUCHECK(hipGetDeviceCount(&n));

	for(int i = 0; i < n; ++i){
		hipDeviceProp_t dev;

		// �f�o�C�X�v���p�e�B�擾
		CUCHECK(hipGetDeviceProperties(&dev, i));

		printf("device %d\n", i);
		printf(" device name : %s\n", dev.name);
		printf(" total global memory : %d (MB)\n", (int)dev.totalGlobalMem/1024/1024);
		printf(" shared memory / block : %d (KB)\n", (int)dev.sharedMemPerBlock/1024);
		printf(" register / block : %d\n", dev.regsPerBlock);
		printf(" warp size : %d\n", dev.warpSize);
		printf(" max pitch : %d (B)\n", (int)dev.memPitch);
		printf(" max threads / block : %d\n", dev.maxThreadsPerBlock);
		printf(" max size of each dim. of block : (%d, %d, %d)\n", dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
		printf(" max size of each dim. of grid  : (%d, %d, %d)\n", dev.maxGridSize[0], dev.maxGridSize[1], dev.maxGridSize[2]);
		printf(" clock rate : %d (MHz)\n", dev.clockRate/1000);
		printf(" total constant memory : %d (KB)\n", (int)dev.totalConstMem/1024);
		printf(" compute capability : %d.%d\n", dev.major, dev.minor);
		printf(" alignment requirement for texture : %d\n", (int)dev.textureAlignment);
		printf(" device overlap : %s\n", (dev.deviceOverlap ? "ok" : "not"));
		printf(" num. of multiprocessors : %d\n", dev.multiProcessorCount);
		printf(" kernel execution timeout : %s\n", (dev.kernelExecTimeoutEnabled ? "on" : "off"));
		printf(" integrated : %s\n", (dev.integrated ? "on" : "off"));
		printf(" host memory mapping : %s\n", (dev.canMapHostMemory ? "on" : "off"));

		printf(" compute mode : ");
		if(dev.computeMode == hipComputeModeDefault) printf("default mode (multiple threads can use) \n");
		else if(dev.computeMode == hipComputeModeExclusive) printf("exclusive mode (only one thread will be able to use)\n");
		else if(dev.computeMode == hipComputeModeProhibited) printf("prohibited mode (no threads can use)\n");

	}

	//printf("Device with Maximum GFLOPS : %d\n", gpuGetMaxGflopsDeviceId());
}

/*!
 * thrust::exclusive_scan�̌Ăяo��
 * @param[out] dScanData scan��̃f�[�^
 * @param[in] dData ���f�[�^
 * @param[in] num �f�[�^��
 */
void CuScan(float* dScanData, float* dData, int num)
{
	thrust::exclusive_scan(thrust::device_ptr<float>(dData),
		thrust::device_ptr<float>(dData + num),
		thrust::device_ptr<float>(dScanData));
}

/*!
 * �f�o�b�O�p : �X�J���[�l���������z��̒l�̕��ϒl�����߂ĕԂ�
 * @param[out] dcol ���q�F�z��(�f�o�C�X������)
 * @param[in] ddens ���q���x�z��(�f�o�C�X������)
 * @param[in] n ���q��
 * @param[in] userparam ���[�U�[�p�����[�^(�C��)
 */
float CuCalAverage(float* data, int n)
{
	if (n == 0) return 0;
	float avg = 0.0f;

	float* data_scan = 0;
	CuAllocateArray((void**)&data_scan, n * sizeof(float));

	// ���v�l�����߂邽�߂�scan(prefix sum)���v�Z
	CuScan(data_scan, data, n);

	// Exclusive scan (�Ō�̗v�f��0�Ԗڂ���n-2�Ԗڂ܂ł̍��v�ɂȂ��Ă���)�Ȃ̂ŁC
	// Scan�O�z��̍Ō�(n-1�Ԗ�)�̗v�f�ƍ��v���邱�ƂŃ|���S�������v�Z
	float lval, lsval;
	CUCHECK(hipMemcpy((void*)&lval, (void*)(data + n - 1), sizeof(float), hipMemcpyDeviceToHost));
	CUCHECK(hipMemcpy((void*)&lsval, (void*)(data_scan + n - 1), sizeof(float), hipMemcpyDeviceToHost));
	float total = lval + lsval;
	avg = total / n;

	if (data_scan != 0) CuFreeArray(data_scan);

	return avg;
}
float CuCalAverageV(float* vdata, int n)
{
	if (n == 0) return 0;
	float avg = 0.0f;

	float* data = 0;
	float* data_scan = 0;
	CuAllocateArray((void**)&data, n * sizeof(float));
	CuAllocateArray((void**)&data_scan, n * sizeof(float));

	dim3 block, grid;
	CuCalGridN(n, block, grid);	// �f�[�^��=�X���b�h���Ƃ��ău���b�N/�O���b�h�T�C�Y���v�Z
	CxVectorToScalar<<<grid, block>>>(vdata, data, n);	// �J�[�l�����s
	hipDeviceSynchronize();

	// ���v�l�����߂邽�߂�scan(prefix sum)���v�Z
	CuScan(data_scan, data, n);

	// Exclusive scan (�Ō�̗v�f��0�Ԗڂ���n-2�Ԗڂ܂ł̍��v�ɂȂ��Ă���)�Ȃ̂ŁC
	// Scan�O�z��̍Ō�(n-1�Ԗ�)�̗v�f�ƍ��v���邱�ƂŃ|���S�������v�Z
	float lval, lsval;
	CUCHECK(hipMemcpy((void*)&lval, (void*)(data + n - 1), sizeof(float), hipMemcpyDeviceToHost));
	CUCHECK(hipMemcpy((void*)&lsval, (void*)(data_scan + n - 1), sizeof(float), hipMemcpyDeviceToHost));
	float total = lval + lsval;
	avg = total / n;

	if (data_scan != 0) CuFreeArray(data_scan);

	return avg;
}

}   // extern "C"
