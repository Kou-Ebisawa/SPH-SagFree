#include "hip/hip_runtime.h"
/*!
  @file sph_kernel.cu

  @brief CUDA : SPH�@
         - CUDA�J�[�l���y�уf�o�C�X�֐����L�q
         - �z�X�g�֐��������ꂽ*.cu�t�@�C������̂݃C���N���[�h(������cpp����C���N���[�h���Ȃ��悤��)

  @author Makoto Fujisawa
  @date 2023-02
 */


//-----------------------------------------------------------------------------
// �C���N���[�h�t�@�C��
//-----------------------------------------------------------------------------
#include "hip/hip_vector_types.h"
#include <hip/hip_math_constants.h>

#include "cuda_utils.h"
#include "cuda_utils.cu"


//-----------------------------------------------------------------------------
// �萔(�f�o�C�X������)
//-----------------------------------------------------------------------------
__device__ __constant__ SceneParameter params;	// �e��p�����[�^


//-----------------------------------------------------------------------------
// device�֐� - �f�o�C�X(GPU)�Ŏ��s�E�f�o�C�X�֐�����̂݌Ăяo����
//-----------------------------------------------------------------------------
/*!
* �O���b�h�ʒu�v�Z
* @param[in] p ���W
* @return �O���b�h���W
*/
__device__
inline int3 calcGridPos(float3 p)
{
    int3 grid;
    grid.x = floor((p.x-params.cell.WorldOrigin.x)/params.cell.CellWidth.x);
    grid.y = floor((p.y-params.cell.WorldOrigin.y)/params.cell.CellWidth.y);
    grid.z = floor((p.z-params.cell.WorldOrigin.z)/params.cell.CellWidth.z);

    grid.x = min(max(grid.x, 0), params.cell.GridSize.x-1);
    grid.y = min(max(grid.y, 0), params.cell.GridSize.y-1);
    grid.z = min(max(grid.z, 0), params.cell.GridSize.z-1);

    return grid;
}

/*!
* �O���b�h���W����1�����z�񒆂ł̈ʒu���v�Z
* @param[in] gridPos �O���b�h���W
* @return �A�h���X
*/
__device__
inline uint calcGridHash(int3 gridPos)
{
    return __umul24(__umul24(gridPos.z, params.cell.GridSize.y), params.cell.GridSize.x)+__umul24(gridPos.y, params.cell.GridSize.x)+gridPos.x;
}

/*!
* ���q�̏Փˏ����m�ِ�
* @param[inout] p,v ���q�ʒu,���x
* @param[in] dt �^�C���X�e�b�v��
*/
__device__
void collision(float3 &p, float3 &v, float dt)
{
    float d;
    float3 nrm, cp;
    float res = params.res;

    // �{�b�N�X�`��̃I�u�W�F�N�g�Ƃ̏Փ�
#if MAX_BOX_NUM
    for(int i = 0; i < params.num_box; ++i){
        if(params.box[i].flg == 0) continue;
        collisionPointBox(p, params.box[i].cen, params.box[i].ext+make_float3(params.particle_radius), params.box[i].rot, params.box[i].inv_rot, cp, d, nrm);
        if(d < 0.0){
            res = (res > 0) ? (res*fabs(d)/(dt*length(v))) : 0.0f;
            v -= (1+res)*nrm*dot(nrm, v);
            p = cp;
        }
    }
#endif

    // ���`��̃I�u�W�F�N�g�Ƃ̏Փ�
#if MAX_SPHERE_NUM
    for(int i = 0; i < params.num_sphere; ++i){
        if(params.sphere[i].flg == 0) continue;
        collisionPointSphere(p, params.sphere[i].cen, params.sphere[i].rad+params.particle_radius, cp, d, nrm);
        if(d < 0.0){
            res = (res > 0) ? (res*fabs(d)/(dt*length(v))) : 0.0f;
            v -= (1+res)*nrm*dot(nrm, v);
            p = cp;
        }
    }
#endif

    // �V�~�����[�V������Ԃ̋��E(AABB)�Ƃ̏Փ�
    float3 l0 = params.boundary_min;
    float3 l1 = params.boundary_max;
    if(distPointAABB(p, 0.5*(l1+l0), 0.5*(l1-l0), cp, d, nrm)){
        res = (res > 0) ? (res*fabs(d)/(dt*length(v))) : 0.0f;
        v -= (1+res)*nrm*dot(nrm, v);
        p = cp;
    }
}


//-----------------------------------------------------------------------------
// global�֐� - �f�o�C�X(GPU)�Ŏ��s�E�z�X�g�֐�����̂݌Ăяo����
//-----------------------------------------------------------------------------
/*!
 * SPH�@�ɂ�闱�q���x�̌v�Z(Poly6�J�[�l��)
 * @param[out] ddens ���q���x
 * @param[in] dvol ���q�̐�
 * @param[in] n ���q��
 */
__global__ 
void CxSphDensity(float*drestdens,float* ddens, float* dvol, float* dmas,int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos0 = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    float m = params.mass;
    float a = params.aw;
    float rest_dens = params.rest_dens;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];
    //�C�V��SPH�ǉ�
    rest_dens = drestdens[sid];

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0-make_float3(h));
    grid_pos1 = calcGridPos(pos0+make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ė��x�v�Z
    float dens = 0.0f;
    for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
        for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
            for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l

                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for(uint j = startIndex; j < endIndex; ++j){
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 rij = pos0-pos1;
                        float r = length(rij);
                        if(r <= h){
                            // Poly6�J�[�l���Ŗ��x���v�Z (rho = �� m Wij)
                            float q = h*h-r*r;

                            float m = params.mass;

                            dens += m*a*q*q*q;
                        }
                    }
                }
            }
        }
    }
    ddens[sid] = dens;
}

/*!
 * ���q�ɓ����͂̌v�Z
 *  - �d�́C���͂Ȃ�
 * @param[out] dpres ���q����
 * @param[in] ddens ���q���x
 * @param[in] n ���q��
 */
__global__ 
void CxSphPressure(float* drestdens,float* dpres, float* ddens, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float p = 0.0f;

    // ���z�C�̂̏�ԕ������Ɋ�Â��v�Z[Muller2003]
    //p = params.gas_k * (ddens[id] - params.rest_dens);

    // Tait�������Ɋ�Â��v�Z:WCSPH[Becker2007]
    float rdens = ddens[id]/params.rest_dens;
    //�C�V��SPH�ǉ�
    rdens = ddens[id] / drestdens[id];

    //p = params.B * (powf(rdens, params.gamma)-1.0f);
    // gamma=7�Œ�̏ꍇ
    p = params.B * (rdens*rdens*rdens*rdens*rdens*rdens*rdens-1.0f);

    // �����̏ꍇ��0�ɂ���
    p = clamp(p, 0.0, 1.0e6);

    // ���q���͂̍X�V(�O���[�o��������������������)
    dpres[id] = p;
}



/*!
* ���q�Q�x�̌v�Z
*  - vorticity confinement�ɂ�闐���\���̂��߂̉Q�x�v�Z
* @param[out] dvort �e���q�̉Q�x�x�N�g��(�f�o�C�X������)
* @param[in] dvel ���q���x
* @param[in] ddens ���q���x
* @param[in] dvol ���q�̐�
* @param[in] datt ���q����(0�ŗ���,1�ŋ��E)
* @param[in] n ���q��
*/
__global__ 
void CxSphVorticity(float* dvort, float* dvel, float* ddens, float* dvol, int* datt, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    uint sid = params.cell.dSortedIndex[id];
    if(datt[sid] != 0){  // ���E���q�̏ꍇ�͉Q�x=0
        float3 v0 = make_float3(0.0f);
        dvort[DIM*sid+0] = v0.x;  dvort[DIM*sid+1] = v0.y; dvort[DIM*sid+2] = v0.z;
        return;
    }

    // ���qi�̕ϐ��l
    float3 pos0 = params.cell.dSortedPos[id];
    float3 vel0 = make_float3(dvel[DIM*sid], dvel[DIM*sid+1], dvel[DIM*sid+2]);

    float h = params.h;
    float m = params.mass;
    float a = params.ag;

    // �p�[�e�B�N�����͂̃O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0-make_float3(h));
    grid_pos1 = calcGridPos(pos0+make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ĉ��͂ɂ���
    float3 f = make_float3(0.0f);
    for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
        for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
            for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);

                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for(uint j = startIndex; j < endIndex; ++j){
                        uint sj = params.cell.dSortedIndex[j];
                        if(sj == sid) continue;

                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 vel1 = make_float3(dvel[DIM*sj], dvel[DIM*sj+1], dvel[DIM*sj+2]);

                        float3 rij = pos0-pos1;
                        float3 vji = vel1-vel0;
                        float dens1 = ddens[sj];

                        float r = length(rij);
                        if(r <= h && r > 0.0001){
                            float q = h-r;
                            float3 gw = a*q*q*rij/r;
                            f += m/dens1*cross(vji, gw);
                        }
                    }
                }
            }
        }
    }

    // ���q�̉Q�x�x�N�g���̍X�V(�O���[�o��������������������)
    dvort[DIM*sid+0] = f.x;
    dvort[DIM*sid+1] = f.y;
    dvort[DIM*sid+2] = f.z;
}


/*!
 * ���q�ɓ����͂̌v�Z
 *  - ���x�����ɂ���悤�Ȉ���
 *  - �d��
 *  - vorticity confinement
 * @param[out] dacc ���q�ɓ�����(�����x)
 * @param[in] dvel ���q���x�z��
 * @param[in] ddens ���q���x
 * @param[in] dpres ���q����
 * @param[in] dvort �e���q�̉Q�x�x�N�g��
 * @param[in] dvol  ���q�̐�
 * @param[in] datt ���q����(0�ŗ���,1�ŋ��E)
 * @param[in] n ���q��
 */
//�C�V��power�ǉ�
//�C�V��dfss�ǉ�
__global__ 
void CxSphForces(float* drestdens,float* dacc, float* dvel, float* ddens, float* dpres, float* dvort, float* dvol,float* dmas, int* datt,float3 power,float* dfss, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    uint sid = params.cell.dSortedIndex[id];
    if(datt[sid] != 0){  // ���E���q�̏ꍇ�͗��q�ɂ������=0
        float3 v0 = make_float3(0.0f);
        dacc[DIM*sid+0] = v0.x;  dacc[DIM*sid+1] = v0.y; dacc[DIM*sid+2] = v0.z;
        return;
    }

    // ���qi�̕ϐ��l
    float3 pos0 = params.cell.dSortedPos[id];
    float3 omega0 = make_float3(dvort[DIM*sid], dvort[DIM*sid+1], dvort[DIM*sid+2]);
    float dens0 = ddens[sid];
    float pres0 = dpres[sid];
    //int3 grid = calcGridPos(pos0);
    float prsi = pres0/(dens0*dens0);

    float h = params.h;
    //float m = params.mass;
    float a = params.ag;

    // �p�[�e�B�N�����͂̃O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0-make_float3(h));
    grid_pos1 = calcGridPos(pos0+make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ĉ��͂ɂ���
    float3 f = make_float3(0.0f);
    float3 eta = make_float3(0.0f);
    for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
        for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
            for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);

                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for(uint j = startIndex; j < endIndex; ++j){
                        uint sj = params.cell.dSortedIndex[j];
                        if(sj == sid) continue;

                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 omega1 = make_float3(dvort[DIM*sj], dvort[DIM*sj+1], dvort[DIM*sj+2]);

                        float3 rij = pos0-pos1;

                        float dens1 = ddens[sj];
                        float pres1 = dpres[sj];
                        float prsj = pres1/(dens1*dens1);

                        float r = length(rij);
                        if(r <= h && r > 0.0001){
                            float q = h-r;
                            float3 gw = a*q*q*rij/r;

                            float m = params.mass;

                            f += -m*(prsi+prsj)*gw; // ���͍��̌v�Z
                            eta += (m/dens1)*length(omega1)*gw;
                        }
                    }
                }
            }
        }
    }
    
    // �d��
    f += params.gravity+power;
    //f += power;

    // Vorticity Confinement
    if(length(eta) > 1e-3){
        f += params.vorticity*dens0*cross(normalize(eta), omega0);
    }

    // ���q�ɂ�����O��(�����x)�̍X�V(�O���[�o��������������������)
    dacc[DIM*sid+0] = f.x; 
    dacc[DIM*sid+1] = f.y;
    dacc[DIM*sid+2] = f.z;
}

/*!
* ���q�ɓ����S���͂̌v�Z
*  - XSPH�łȂ��S�������(�����x)�Ƃ��Čv�Z������@[Becker2007]
* @param[inout] dacc ���q�ɓ�����(�����x)
* @param[in] dvel ���q���x
* @param[in] ddens ���q���x
* @param[in] dvol ���q�̐�
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)
* @param[in] n ���q��
*/
__global__ 
void CxSphViscosity(float* drestdens,float* dacc, float* dvel, float* ddens, float* dvol,float* dmas, int* datt, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    uint sid = params.cell.dSortedIndex[id];
    if(datt[sid] != 0) return;

    // ���qi�̕ϐ��l
    float3 pos0 = params.cell.dSortedPos[id];
    float3 vel0 = make_float3(dvel[DIM*sid], dvel[DIM*sid+1], dvel[DIM*sid+2]);
    float dens0 = ddens[sid];

    float h = params.h;
    float a = params.ag;
    float alpha = params.viscosity;  // �S���萔�D�_������[0.08, 0.5]�ƍs���Ă��邪���ꂾ�Ƒ傫������...
    float cs = 88.5;
    float eps = 0.001*h*h;
    float rest_dens = params.rest_dens;
    //�C�V��SPH�ǉ�
    rest_dens = drestdens[sid];

    // �p�[�e�B�N�����͂̃O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0-make_float3(h));
    grid_pos1 = calcGridPos(pos0+make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ĉ��͂ɂ���
    float3 f = make_float3(0.0f);
    for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
        for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
            for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);

                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for(uint j = startIndex; j < endIndex; ++j){
                        uint sj = params.cell.dSortedIndex[j];
                        if(sj == sid) continue;

                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 vel1 = make_float3(dvel[DIM*sj], dvel[DIM*sj+1], dvel[DIM*sj+2]);
                        float3 rij = pos0-pos1;

                        float dens1 = ddens[sj];
                        float m = dmas[j];
                        float vx = dot(vel0-vel1, rij);

                        float r = length(rij);
                        if(r <= h && r > 0.0001 && vx < 0.0f){
                            float nu = (2.0f*alpha*h*cs)/(dens0+dens1);
                            float visc = -nu*(vx/(r*r+eps));
                            float q = h-r;
                            f += -m*visc*a*q*q*rij/r;
                        }
                    }
                }
            }
        }
    }

    // ���q�ɂ�����O��(�����x)�̍X�V(�O���[�o��������������������)
    dacc[DIM*sid+0] += f.x; 
    dacc[DIM*sid+1] += f.y;
    dacc[DIM*sid+2] += f.z;
}

/*!
* ���q�ɓ����͂̌v�Z
*  - XSPH Artificial Viscosity�ɂ�鑬�x�X�V[Schechter2012]
* @param[inout] dvel ���q���x
* @param[in] ddens ���q���x
* @param[in] dvol ���q�̐�
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)
* @param[in] n ���q��
*/
__global__ 
void CxSphXSPHViscosity(float* dvel, float* ddens, float* dvol,float* dmas, int* datt, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    uint sid = params.cell.dSortedIndex[id];
    if(datt[sid] != 0) return;

    // ���qi�̕ϐ��l
    float3 pos0 = params.cell.dSortedPos[id];
    float3 vel0 = make_float3(dvel[DIM*sid], dvel[DIM*sid+1], dvel[DIM*sid+2]);

    float h = params.h;
    float a = params.aw;
    float eps = params.viscosity;
    float rest_dens = params.rest_dens;

    // �p�[�e�B�N�����͂̃O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0-make_float3(h));
    grid_pos1 = calcGridPos(pos0+make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ĉ��͂ɂ���
    float3 dv = make_float3(0.0f);
    for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
        for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
            for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);

                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for(uint j = startIndex; j < endIndex; ++j){
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 vel1 = make_float3(dvel[DIM*sj], dvel[DIM*sj+1], dvel[DIM*sj+2]);
                        float3 rij = pos0-pos1;

                        float dens1 = ddens[sj];
                        float m = dmas[sj];

                        float r = length(rij);
                        if(r <= h && r > 0.0001){
                            float q = h*h-r*r;
                            //float m = rest_dens*dvol[sj];
                            dv += (m/dens1)*(vel1-vel0)*a*q*q*q;
                        }
                    }
                }
            }
        }
    }

    dv *= eps;

    // ���q���x�̍X�V(�O���[�o��������������������)
    dvel[DIM*sid+0] += dv.x; 
    dvel[DIM*sid+1] += dv.y;
    dvel[DIM*sid+2] += dv.z;
}


/*!
 * ���q��O�i�I�C���[�@�ňړ�������
 *  - �ʒu�̑��x�ɂ��ϕ�
 *  - ���E�������܂�
 * @param[inout] dpos ���q�ʒu
 * @param[inout] dvel ���q���x
 * @param[in] dacc ���q�ɓ�����(�����x)
 * @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)
 * fix:�C�V��ǉ� 1�Ȃ�ΌŒ�_
* @param[in] n ���q��
 */
__global__ 
void CxSphIntegrate(float* dpos, float* dvel, float* dacc, int* datt,int* dfix, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)
    if(datt[id] != 0){  // ���E���q�̏ꍇ�͑��x��0�ɂ��Ĉʒu�͕ς��Ȃ�
        float3 v0 = make_float3(0.0f);
        dvel[DIM*id+0] = v0.x;  dvel[DIM*id+1] = v0.y; dvel[DIM*id+2] = v0.z;
        return;
    }
    //�C�V��ǉ�
    //�Œ�_�Ȃ�΁A���x��0�ɂ��ď������X�L�b�v
    if (dfix[id] == 1||id==0 || dfix[id - 1] == 1) {// 
        float3 v0 = make_float3(0.0f);
        dvel[DIM * id + 0] = v0.x;  dvel[DIM * id + 1] = v0.y; dvel[DIM * id + 2] = v0.z;
        return;
    }
    // ���q�ʒu,���x,��
    float3 p = make_float3(dpos[DIM*id+0], dpos[DIM*id+1], dpos[DIM*id+2]);
    float3 v = make_float3(dvel[DIM*id+0], dvel[DIM*id+1], dvel[DIM*id+2]);
    float3 a = make_float3(dacc[DIM*id+0], dacc[DIM*id+1], dacc[DIM*id+2]);
    float dt = params.dt;

    // �X�V�ʒu�C���x�̍X�V
    v += dt*a;
    p += dt*v;

    // ���͋��E�Ƃ̏Փˏ���
    collision(p, v, dt);

    // ���q�ʒu�E���x�̍X�V(�O���[�o��������������������)
    dpos[DIM*id+0] = p.x;  dpos[DIM*id+1] = p.y; dpos[DIM*id+2] = p.z;
    dvel[DIM*id+0] = v.x;  dvel[DIM*id+1] = v.y; dvel[DIM*id+2] = v.z;
}

/*!
* ���q��O�i�I�C���[�@�ňړ�������
*  - ���x�̎��Ԑϕ��̂�, XSPH�̂݁C���E�����Ȃ�
* @param[inout] dvel ���q���x
* @param[in] dacc ���q�ɓ�����(�����x)
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)
* @param[in] n ���q��
*/
__global__ 
void CxSphIntegrateVelocity(float* dvel, float* dacc, int* datt, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)
    if(datt[id] != 0){  // ���E���q�̏ꍇ�͑��x��0�ɂ��Ĉʒu�͕ς��Ȃ�
        float3 v0 = make_float3(0.0f);
        dvel[DIM*id+0] = v0.x;  dvel[DIM*id+1] = v0.y; dvel[DIM*id+2] = v0.z;
        return;
    }

    // ���q�ʒu,���x,��
    float3 v = make_float3(dvel[DIM*id+0], dvel[DIM*id+1], dvel[DIM*id+2]);
    float3 a = make_float3(dacc[DIM*id+0], dacc[DIM*id+1], dacc[DIM*id+2]);
    float dt = params.dt;

    // �X�V�ʒu�C���x�̍X�V
    v += dt*a;

    // ���q�ʒu�E���x�̍X�V(�O���[�o��������������������)
    dvel[DIM*id+0] = v.x;  dvel[DIM*id+1] = v.y; dvel[DIM*id+2] = v.z;
}
/*!
* ���q��O�i�I�C���[�@�ňړ�������
*  - �ʒu�̑��x�ɂ��ϕ��̂݁CXSPH�p�C���E�������܂�
* @param[inout] dpos ���q�ʒu
* @param[inout] dvel ���q���x
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)
* @param[in] n ���q��
*/
//�C�V�� dfix�ǉ�
__global__ 
void CxSphIntegratePosition(float* dpos, float* dvel, int* datt,int* dfix, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n || datt[id] != 0) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    //�C�V��ǉ�
    //�Œ�_�Ȃ�΁A���x��0�ɂ��ď������X�L�b�v
    if (dfix[id] == 1 || id == 0 || dfix[id - 1] == 1) { 
        float3 v0 = make_float3(0.0f);
        dvel[DIM * id + 0] = v0.x;  dvel[DIM * id + 1] = v0.y; dvel[DIM * id + 2] = v0.z;
        return;
    }

    // ���q�ʒu,���x,��
    float3 p = make_float3(dpos[DIM*id+0], dpos[DIM*id+1], dpos[DIM*id+2]);
    float3 v = make_float3(dvel[DIM*id+0], dvel[DIM*id+1], dvel[DIM*id+2]);
    float dt = params.dt;

    // �X�V�ʒu�C���x�̍X�V
    p += dt*v;

    // ���͋��E�Ƃ̏Փˏ���
    collision(p, v, dt);

    // ���q�ʒu�E���x�̍X�V(�O���[�o��������������������)
    dpos[DIM*id+0] = p.x;  dpos[DIM*id+1] = p.y; dpos[DIM*id+2] = p.z;
    dvel[DIM*id+0] = v.x;  dvel[DIM*id+1] = v.y; dvel[DIM*id+2] = v.z;
}


/*!
* ���E���q�����̂��߂̗��q�̐όv�Z
*  - ���E���q�� "Versatile Rigid-Fluid Coupling for Incompressible SPH", 2.2 ��(3)�̏��V_bi �Ōv�Z
*  - ���̗��q�� V=mass/rest_dens
* @param[out] dvol ���q�̐�
* @param[in] datt ���q����(0�ŗ��̗��q�C����ȊO�ŋ��E���q)
* @param[in] n �������闱�q��(offset����̑��ΓI�Ȉʒu)
* @param[in] v ���̗��q�̏ꍇ�̗��q�̐ϒl
*/
__global__ 
void CxSphCalVolume(float* dvol, int *datt, int n, float v)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    uint sid = params.cell.dSortedIndex[id];
    int att = datt[sid];
    if(att == 0){   // ���̗��q�̏ꍇ
        dvol[sid] = v;
        return;
    }

    float3 pos0 = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    float m = params.mass;
    float a = params.aw;

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0-make_float3(h));
    grid_pos1 = calcGridPos(pos0+make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����đ̐όv�Z
    float mw = 0.0f;    // ��mW
    for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
        for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
            for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l
                uint startIndex = params.cell.dCellStart[ghash];                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for(uint j = startIndex; j < endIndex; ++j){
                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 rij = pos0-pos1;
                        float r = length(rij);
                        if(r <= h){
                            float q = h*h-r*r;
                            mw += m*a*q*q*q;
                        }
                    }
                }
            }
        }
    }

    // �v�Z�����̐ς��O���[�o���������ɏ�������
    dvol[sid] = m/mw;
}


/*!
* �O���b�h��ł̖��x���v�Z(�\�ʃ��b�V�������p)
* @param[out] dF ���x�l���i�[����O���b�h�Z���z��
* @param[in] dvol ���q�̐�
* @param[in] datt ���q����(0�ŗ���,1�ŋ��E)
* @param[in] n ���q��
* @param[in] gnum �O���b�h��
* @param[in] gmin �O���b�h�ŏ����W
* @param[in] glen �O���b�h��
*/
__global__
void CxSphDensityAtCell(float* dF, float* dvol, int* datt, int n, 
                        int3 gnum, float3 gmin, float3 glen)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int3 gridPos = calcGridPos(id, gnum);

    if(gridPos.x < gnum.x && gridPos.y < gnum.y && gridPos.z < gnum.z){
        float3 pos0;    // �O���b�h�Z�����S���W
        pos0.x = gmin.x+(gridPos.x)*glen.x;
        pos0.y = gmin.y+(gridPos.y)*glen.y;
        pos0.z = gmin.z+(gridPos.z)*glen.z;

        float h = params.effective_radius;
        float m = params.mass;
        float a = params.aw;

        int3 grid_pos0, grid_pos1;
        grid_pos0 = calcGridPos(pos0-make_float3(h));
        grid_pos1 = calcGridPos(pos0+make_float3(h));

        float dens = 0.0f;
        for(int z = grid_pos0.z; z <= grid_pos1.z; ++z){
            for(int y = grid_pos0.y; y <= grid_pos1.y; ++y){
                for(int x = grid_pos0.x; x <= grid_pos1.x; ++x){
                    int3 ngrid = make_int3(x, y, z);
                    uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l

                    // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                    uint startIndex = params.cell.dCellStart[ghash];
                    if(startIndex != 0xffffffff){	// �Z������łȂ����̃`�F�b�N
                        // �Z�����̃p�[�e�B�N���Ŕ���
                        uint endIndex = params.cell.dCellEnd[ghash];
                        for(uint j = startIndex; j < endIndex; ++j){
                            uint sj = params.cell.dSortedIndex[j];
                            if(datt[sj] != 0) continue; // ���E���q�͕\�ʃ��b�V�������ɂ͎g��Ȃ�
                            float3 pos1 = params.cell.dSortedPos[j];
                            float3 rij = pos0-pos1;
                            float r = length(rij);
                            if(r <= h){
                                // Poly6�J�[�l���Ŗ��x���v�Z (rho = �� m Wij)
                                float q = h*h-r*r;
                                dens += m*a*q*q*q;
                            }
                        }
                    }
                }
            }
        }

        dF[gridPos.x+gridPos.y*gnum.x+gridPos.z*gnum.x*gnum.y] = dens;
    }

}


//-----------------------------------------------------------------------------
// ���q�̕`��F�̌v�Z
//-----------------------------------------------------------------------------
/*!
* ���q�̕`��F�𗱎q�̎������ʂ���v�Z
* @param[out] dcol ���q�F�z��(�f�o�C�X������)
* @param[in] dval ���q�����ʔz��(�f�o�C�X������)
* @param[in] n ���q��
*/
__global__ 
void CxColorScalar(float* dcol, int* datt, float* dval, int n, float3 c1, float3 c2, float2 range)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    // �����ʂƑ���
    float val = dval[id];
    int att = datt[id];

    // ���q�F�̌v�Z
    float l = range.y-range.x;
    float t = clamp((val-range.x)/l, 0.0f, 1.0f);
    float3 col = lerp(c1, c2, t);

    // ���q�F�̃O���[�o���������ւ̊i�[
    dcol[4*id+0] = col.x;  dcol[4*id+1] = col.y; dcol[4*id+2] = col.z;
    dcol[4*id+3] = (1.0f-(float)(att));
}
__global__ 
void CxColorVector(float* dcol, int* datt, float* dval, int n, float3 c1, float3 c2, float2 range)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    // �����ʂƑ���
    float val = length(make_float3(dval[DIM*id], dval[DIM*id+1], dval[DIM*id+2]));
    int att = datt[id];

    // ���q�F�̌v�Z
    float l = range.y-range.x;
    float t = clamp((val-range.x)/l, 0.0f, 1.0f);
    float3 col = lerp(c1, c2, t);

    // ���q�F�̃O���[�o���������ւ̊i�[
    dcol[4*id+0] = col.x;  dcol[4*id+1] = col.y; dcol[4*id+2] = col.z;
    dcol[4*id+3] = (1.0f-(float)(att));
}

/*!
* ���q�̕`��F�ݒ� : ���ׂē����F
* @param[out] dcol ���q�F�z��(�f�o�C�X������)
* @param[in] ddens ���q���x�z��(�f�o�C�X������)
* @param[in] n ���q��
*/
__global__ 
void CxColorConstant(float* dcol, int* datt, float3 col, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)
    
    // ���q����
    int att = datt[id];

    // ���q�F�̃O���[�o���������ւ̊i�[
    dcol[4*id+0] = col.x;  dcol[4*id+1] = col.y; dcol[4*id+2] = col.z;
    dcol[4*id+3] = (1.0f-(float)(att));
}

/*!
* �f�o�b�O�p : �x�N�g���z�񂩂�x�N�g���̑傫��(�X�J���[�l)�̔z����v�Z
* @param[in] dvdata �x�N�g���l�z��(�f�o�C�X������)
* @param[out] dsdata �X�J���[�l�z��(�f�o�C�X������)
* @param[in] n ���q��
*/
__global__ 
void CxVectorToScalar(float* dvdata, float* dsdata, int n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    // �x�N�g���̑傫�������߂�
    float val = length(make_float3(dvdata[DIM*id], dvdata[DIM*id+1], dvdata[DIM*id+2]));

    // ���ʂ̃O���[�o���������ւ̊i�[
    dsdata[id] = val;
}


//-----------------------------------------------------------------------------
// �n�b�V��
//-----------------------------------------------------------------------------
/*!
 * �e���q�̃O���b�h�n�b�V���l�v�Z
 * @param[out] dhash �e���q�̃O���b�h�n�b�V���l���i�[�����z��
 * @param[out] dsortedidx �e���q�̃C���f�b�N�X���i�[�����z��(�ォ��n�b�V���l�Ń\�[�g����� -> �����_�ł͂܂��\�[�h�ς݂ł͂Ȃ�)
 * @param[in] dpos ���q�ʒu���i�[�����z��
 * @param[in] n ���q��
 */
__global__
void CxCalcHash(uint* dhash, uint* dsortedidx, float* dpos, uint n)
{
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    // ���q�ʒu
    float3 p = make_float3(dpos[DIM*id+0], dpos[DIM*id+1], dpos[DIM*id+2]);

    // ���q�ʒu����܂܂��O���b�h�Z���̃n�b�V���l���v�Z
    int3 grid = calcGridPos(p);
    uint hash = calcGridHash(grid);

    dhash[id] = hash;
    dsortedidx[id] = id;
}

/*!
 * �p�[�e�B�N���f�[�^���\�[�g���āC�n�b�V�����̊e�Z���̍ŏ��̃A�h���X������
 * @param[in] cell �p�[�e�B�N���O���b�h�f�[�^
 * @param[in] dpos ���q�ʒu�z��
 * @param[in] dvel ���q���x�z��
 */
__global__
void CxReorderDataAndFindCellStartD(Cell cell, float* dpos, float* dvel, uint n)
{
    // �V�F�A�[�h������
    extern __shared__ uint sharedHash[];	// �T�C�Y : blockSize+1

    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    uint hash;
    if(id < n){ // �V�F�A�[�h�������g�p�̂��߂ɃX���b�h�������s���̂�(id >= n)�̎���return�͂��Ȃ�
        hash = cell.dGridParticleHash[id];	// �n�b�V���l
        sharedHash[threadIdx.x+1] = hash;	// �n�b�V���l���V�F�A�[�h�������Ɋi�[

        if(id > 0 && threadIdx.x == 0){
            // �e�V�F�A�[�h�������̍ŏ��ׂ͗̃O���b�h�̃p�[�e�B�N���̃n�b�V���l���i�[
            sharedHash[0] = cell.dGridParticleHash[id-1];
        }
    }

    __syncthreads();    // �X���b�h����(���̃X���b�h���V�F�A�[�h�������i�[�������I����܂ő҂�)

    if(id < n){
        // �C���f�b�N�X0�ł���C�������́C��O�̃p�[�e�B�N���̃O���b�h�n�b�V���l���قȂ�ꍇ�C
        // �p�[�e�B�N���͕����̈�̍ŏ�
        if(id == 0 || hash != sharedHash[threadIdx.x]){
            cell.dCellStart[hash] = id;
            if(id > 0){
                // ��O�̃p�[�e�B�N���́C��O�̕����̈�̍Ō�
                cell.dCellEnd[sharedHash[threadIdx.x]] = id;
            }
        }

        // �C���f�b�N�X���Ō�Ȃ�΁C�����̈�̍Ō�
        if(id == cell.uNumParticles-1){
            cell.dCellEnd[hash] = id+1;
        }

        // �ʒu�Ƒ��x�̃f�[�^����ёւ�
        // �\�[�g�����C���f�b�N�X�ŎQ�Ƃ��\�����T�����̃O���[�o���������A�N�Z�X���ɗ͗}���邽�߂Ƀf�[�^���̂��̂���ёւ���
        uint sid = cell.dSortedIndex[id];
        float3 pos = make_float3(dpos[DIM*sid+0], dpos[DIM*sid+1], dpos[DIM*sid+2]);
        float3 vel = make_float3(dvel[DIM*sid+0], dvel[DIM*sid+1], dvel[DIM*sid+2]);

        cell.dSortedPos[id] = pos;
        cell.dSortedVel[id] = vel;
    }
}

//---------------------------------------------
//�ȉ��A�C�V��ǉ� �l������float4(x,y,z,w)�Ƃ��Ĉ���(�o������Q�l)

//�o������̃R�[�h����
//��̎l�����̐ς����
__host__ __device__ float4 quatProduct(float4 a, float4 b) {
    return make_float4(
        a.x * b.w + a.w * b.x - a.z * b.y + a.y * b.z,
        a.y * b.w + a.z * b.x + a.w * b.y - a.x * b.z,
        a.z * b.w - a.y * b.x + a.x * b.y + a.w * b.z,
        a.w * b.w - a.x * b.x - a.y * b.y - a.z * b.z
    );
}

//�o������̃R�[�h����
//�l�����̋��������
__host__ __device__ float4 quatConjugate(float4 quat) {
    return make_float4(-quat.x, -quat.y, -quat.z, quat.w);
}

//�o������̃R�[�h����
//3�����x�N�g���Ǝl�����̐ς����
__host__ __device__ float3 rotVecByQuat(float3 vec, float4 quat) {
    float4 vecq = make_float4(vec, 0.0f);
    float4 vecq_dash = quatProduct(quatProduct(quat, vecq), quatConjugate(quat));
    return make_float3(vecq_dash);
};

//3�����x�N�g���̒��������߂�
__host__ __device__ float Length(float3 vec) {
    return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

//4�����x�N�g���̒��������߂�
__host__ __device__ float Length(float4 vec) {
    return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z + vec.w * vec.w);
}

//XPBD�ɗp����ɂ�S��0�ɂ���
//dlamb_ss:�L�ѐ���̃�
//dlamb_bt:�Ȃ�����̃�
__global__
void CxSetLambdaZero(float* dlamb_ss,float* dlamb_bt,int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    dlamb_ss[DIM * id] = dlamb_ss[DIM * id + 1] = dlamb_ss[DIM * id + 2] = 0.f;
    dlamb_bt[QUAT * id] = dlamb_bt[QUAT * id + 1] = dlamb_bt[QUAT * id + 2] = dlamb_bt[QUAT * id + 3] = 0.f;
}

//XPBD�̐L�сE����f����
//dpos:�ʒu
//dcurpos:�ʒu�X�V�O�̈ʒu(�����p�ɑ��x���l����ꍇ)
//dmas:����
//dlen:���
//dkss:�Ȃ�����
//dquat:�p��
//dcurquat:�X�V�O�̎p��(�����p�Ɋp���x���l����ꍇ)
//dlamb_ss:XPBD�̃�
//fix:�Œ�_���ǂ���(1�Ȃ�ΌŒ�_)
//dt:�^�C���X�e�b�v
//n:���q��
//odd_even:�����̃X���b�hID����̃X���b�hID���𔻕�
//example_flag:�`��ɂ���āC�������ꕔ�ύX����
__global__
void CxStretchingShearConstraint(float* dpos,float* dcurpos, float* dmas, float* dlen, float* dkss, float* dquat,float* dcurquat, float* dlamb_ss, int* dfix, float dt, int n,int odd_even,int iter,bool example_flag) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    id = id * 2 + odd_even;
    
    if (id >= n-1) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    if (dfix[id] == 1 || dfix[id + 1] == 1) return;//�e���̖��ɕӂ̐������s�� 

    float3 pos0 = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 pos1 = make_float3(dpos[DIM * (id + 1)], dpos[DIM * (id + 1) + 1], dpos[DIM * (id + 1) + 2]);

    float mass = dmas[id];//���݂͗��q�̏d���͑S�ē������Ƃ��Ă���
    float length = dlen[id];
    float kss = dkss[id];
    float4 quat = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);//�l������(x,y,z,w)�ɂȂ�悤�Ɏ󂯎��
    float3 lambda_ss = make_float3(dlamb_ss[DIM * id], dlamb_ss[DIM * id + 1], dlamb_ss[DIM * id + 2]);

    float w1 = 1 / mass;//�d��
    float w2 = 1 / mass;//�d��
    float wq = 1.0;//�ӂ̏d��
    //�d�݂̓K�؂Ȑݒ�ɂ��Ă͂܂���܂��Ă��Ȃ�
    //�d�݂�2�̒��_�̎��ʂ̘a�Ƃ��Đݒ�
    wq = (w1 + w2) ;
    //�����𗘗p���Ă݂�
    //wq = (w1 + w2) / length / length * 4;
    //wq = w1 * w2 / (w1 + w2);
    //wq = 7.5e3;
    //Example Rod
    if (example_flag) {
        w1 = w2 = 1/length;
        wq = 1.0e5*length;
    }

    //�Œ�_�̏ꍇ�C�d�݂�0�ɂ���
    if (dfix[id-1] == 1) {
        w1 = 0;
        wq = 0;
    }

    float alpha = 1 / (kss * dt * dt);
    float3 ds3 = rotVecByQuat(make_float3(0.f, 0.f, 1.f), quat);//�p���̊�ƂȂ�x�N�g����z������O��Ƃ��Ă���

    //�������l����ꍇ-----------------------------
    //float damping = 0.1;//�����W��
    //float beta = damping * dt * dt;
    //float gamma = alpha * beta / dt;
    //float weight_with_damping = (1 + gamma) * ((1 / (length * length) * (w1 + w2)) + 4 * wq) + alpha;//(1+gamma)*(\nablaC^2 w1+\nablaC^2 w2+\nablaC^2 wq)+alpha(�ꕔ�ȗ������ăR�����g)

    //float3 curpos0 = make_float3(dcurpos[DIM * id], dcurpos[DIM * id + 1], dcurpos[DIM * id + 2]);
    //float3 curpos1 = make_float3(dcurpos[DIM * (id + 1)], dcurpos[DIM * (id + 1) + 1], dcurpos[DIM * (id + 1) + 2]);
    //float4 curquat = make_float4(dcurquat[QUAT * id], dcurquat[QUAT * id + 1], dcurquat[QUAT * id + 2], dcurquat[QUAT * id + 3]);

    //float3 tmp_v0 = pos0 - curpos0;
    //float3 tmp_v1 = pos1 - curpos1;

    //float3 tmp_angvel = 2.f * make_float3(quatProduct(quatConjugate(curquat), quat));
    //float4 nablaC_q = -2.f * quatProduct(quat, make_float4(0.f, 0.f, -1.f, 0.f));//q_s*\bar{e_3}

    //float3 sum = tmp_v0 * (-1 / length) + tmp_v1 * (1 / length) + make_float3(quatProduct(nablaC_q,make_float4(tmp_angvel,0.f)));//\nablaC*v���v�Z�C�ŏI�e��\nablaC*(x_i-x^n)���l�����ɒu��������

    //float3 mole_with_damping = (pos1 - pos0) / length - ds3 + alpha * lambda_ss + gamma * sum;//�}�C�i�X�������Ă��Ȃ�
    //---------------------------------------------

    float weight = w1 + w2 + length * length * (4 * wq + alpha);//����
    float3 mole = length * (pos0 - pos1 + length * ds3 - alpha * length * lambda_ss);//���q
    float3 lambda = mole / weight;//����

    //�������l����ꍇ------------------------------------------
    //lambda = -mole_with_damping / weight_with_damping;
    //----------------------------------------------------------

    //�d�ݒǉ�
    float3 delta_pos0 = -w1*lambda / length;//��x0(�_���ƕ����t)
    float3 delta_pos1 = w2*lambda / length;//��x1(�_���ƕ����t
    float4 q_e3_bar = quatProduct(quat, make_float4(0.f, 0.f, -1.f, 0.f));
    
    float4 inter_quat = quatProduct(make_float4(lambda, 0.f), q_e3_bar);
    float4 delta_quat = -wq * 2.f * inter_quat;
    float4 new_quat = quat + delta_quat;//�X�V���qs
    new_quat = normalize(new_quat);

    //�ʒu�X�V
    if (dfix[id-1] == 0) {
        dpos[id * DIM] += delta_pos0.x;
        dpos[id * DIM + 1] += delta_pos0.y;
        dpos[id * DIM + 2] += delta_pos0.z;
    }

    dpos[(id + 1) * DIM] += delta_pos1.x;
    dpos[(id + 1) * DIM + 1] += delta_pos1.y;
    dpos[(id + 1) * DIM + 2] += delta_pos1.z;
    
    //�p���̍X�V
    if (dfix[id] == 0) {
        dquat[id * QUAT] = new_quat.x;
        dquat[id * QUAT + 1] = new_quat.y;
        dquat[id * QUAT + 2] = new_quat.z;
        dquat[id * QUAT + 3] = new_quat.w;
    }

    //XPBD�ł̃ɂ̐ݒ�
    dlamb_ss[id * DIM] += lambda.x;
    dlamb_ss[id * DIM + 1] += lambda.y;
    dlamb_ss[id * DIM + 2] += lambda.z;
}

//�Ȃ��˂��ꐧ��̕���������
//delta_omega=cur_omega-rest_omega
//delta_omega_plus=cur_omega+rest_omega
__host__ __device__ 
int deltaOmegaSign(float4 delta_omega, float4 delta_omega_plus) {
    if (dot(delta_omega, delta_omega) > dot(delta_omega_plus, delta_omega_plus)) return -1;

    return 1;
}

//�Ȃ��˂��ꐧ��̒ǉ�
//dmas:����
//dquat:�ӂ̎p��(�l����)
//dcurquat:�X�V�O�̎p��(�����p�ɑ��x���l����)
//domega:��_���{�[�x�N�g��
//dkbt:�Ȃ�����
//dlamb_bt:�Ȃ�����ɗp�����
//dfix:�Œ�_���ǂ�����\��(1�Ȃ�ΌŒ�_)
//dt:�^�C���X�e�b�v
//n:���q��
//odd_even:�����̃X���b�hID����̃X���b�hID���𔻕�
//example_flag:�`��ɂ���āC�������ꕔ�ύX����
__global__ 
void CxBendTwistConstraint(float* dmas,float* dquat,float* dcurquat, float* domega, float* dkbt, float* dlamb_bt, float* dlength, int* dfix, float dt, int n, int odd_even, int iter,bool example_flag) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    id = id * 2 + odd_even;
    if (id >= n - 2) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō��2���q�̓X�L�b�v
    if (dfix[id + 1] == 1 || dfix[id + 2] == 1) return;//���[�̗��q�Ƃ��̂ЂƂO�̗��q�ł́A�G�b�W������Ȃ�

    float kbt = dkbt[id];
    float4 quat1 = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);
    float4 quat2 = make_float4(dquat[QUAT * (id + 1)], dquat[QUAT * (id + 1) + 1], dquat[QUAT * (id + 1) + 2], dquat[QUAT * (id + 1) + 3]);
    float4 rest_omega = make_float4(domega[QUAT * id], domega[QUAT * id + 1], domega[QUAT * id + 2], domega[QUAT * id + 3]);
    float4 lambda_bt = make_float4(dlamb_bt[QUAT * id], dlamb_bt[QUAT * id + 1], dlamb_bt[QUAT * id + 2], dlamb_bt[QUAT * id + 3]);

    float dlen1 = dlength[id];
    float dlen2 = dlength[id + 1];

    float wq1 = 1.0f;//1/(1.0e-3)*dlen1
    float wq2 = 1.0f;
    float alpha = 1 / (kbt * dt * dt);
    //�d�݂̐ݒ�ɗ��p(�S�Ă̎��ʂ͓������Ɖ���)
    float mass = dmas[id];
    //�d�݂̓K�؂Ȑݒ�͂܂���܂��Ă��Ȃ�
    //wq1 = 1.0f / dlen1;
    //wq2 = 1.0f / dlen2;
    wq1 = wq2 = 2 / mass * 10.f;//*10.f
    //wq1 = 2 / mass / dlen1 / dlen1 * 4;
    //wq2 = 2 / mass / dlen2 / dlen2 * 4;
    if (example_flag) {
        wq1 = wq2 = 1.0e5 * dlen1;
    }

    //�Œ肷��G�b�W�Ȃ�d�݂�0�ɂ���
    if (dfix[id] == 1) {
        wq1 = 0;
    }

    float weight = wq1 + wq2 + alpha;//����
    float4 cur_omega = quatProduct(quatConjugate(quat1), quat2);
    int s = deltaOmegaSign(cur_omega - rest_omega, cur_omega + rest_omega);//���������߂�

    //�������l����ꍇ-------------------------------------------------
    //float damping = 0.05;//�����W��
    //float beta = damping * dt * dt;
    //float gamma = alpha * beta / dt;
    //float weight_with_damping = (1 + gamma) * (wq1 + wq2) + alpha;

    //float4 curquat1 = make_float4(dcurquat[QUAT * id], dcurquat[QUAT * id + 1], dcurquat[QUAT * id + 2], dcurquat[QUAT * id + 3]);
    //float4 curquat2 = make_float4(dcurquat[QUAT * (id + 1)], dcurquat[QUAT * (id + 1) + 1], dcurquat[QUAT * (id + 1) + 2], dcurquat[QUAT * (id + 1) + 3]);

    //float4 tmp_angvel1 = 2.f * quatProduct(quatConjugate(curquat1), quat1);
    //float4 tmp_angvel2 = 2.f * quatProduct(quatConjugate(curquat2), quat2);

    //float4 sum = tmp_angvel1 * (-quat2) + tmp_angvel2 * quat1;

    //float4 mole_with_damping = -(cur_omega - s * rest_omega) - alpha * lambda_bt - gamma * sum;//�}�C�i�X�����łɂ����Ă���
    //-----------------------------------------------------------------

    float4 delta_omega = cur_omega - (s * rest_omega);
    //delta_omega.w = 0.f;//omega.w=0�Ƃ��Ă���
    float4 lambda = (-delta_omega - alpha * lambda_bt) / weight;
    //lambda.w = 0.f;

    //�������l����ꍇ------------------------------------------------
    //lambda = mole_with_damping / weight_with_damping;
    //----------------------------------------------------------------

    //�d�ݒǉ�
    float4 delta_quat1 = wq1 * quatProduct(quat2, quatConjugate(lambda));
    float4 delta_quat2 = wq2 * quatProduct(quat1, lambda);
    float4 new_quat1 = normalize(quat1 + delta_quat1);
    float4 new_quat2 = normalize(quat2 + delta_quat2);

    //quat1�̍X�V
    if (dfix[id] == 0) {
        dquat[QUAT * id] = new_quat1.x;
        dquat[QUAT * id + 1] = new_quat1.y;
        dquat[QUAT * id + 2] = new_quat1.z;
        dquat[QUAT * id + 3] = new_quat1.w;
    }
    //quat2�̍X�V
    dquat[QUAT * (id + 1)] = new_quat2.x;
    dquat[QUAT * (id + 1) + 1] = new_quat2.y;
    dquat[QUAT * (id + 1) + 2] = new_quat2.z;
    dquat[QUAT * (id + 1) + 3] = new_quat2.w;

    //lambda�̍X�V
    dlamb_bt[QUAT * id] += lambda.x;
    dlamb_bt[QUAT * id + 1] += lambda.y;
    dlamb_bt[QUAT * id + 2] += lambda.z;
    dlamb_bt[QUAT * id + 3] += lambda.w;
}

//�Փː���̎���
//dpos:�ʒu
//dvel:���x
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//center:�є��Ƃ̏Փ˂������������̒��S
//rad:�є��Ƃ̏Փ˂������������̔��a
//dt:�^�C���X�e�b�v
//n:���q��
__global__
void CxCollisionConstraint(float* dpos, float* dvel, int* dfix, float3 center, float rad, float dt, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n)return;
    if (dfix[id] == 1) return;

    float3 pos = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 vel = make_float3(dvel[DIM * id], dvel[DIM * id + 1], dvel[DIM * id + 2]);

    float3 d = pos - center;
    float l = rad - length(d);
    if (l <= 0) return;

    float3 norm = normalize(d);

    dpos[DIM * id] += l * norm.x;
    dpos[DIM * id + 1] += l * norm.y;
    dpos[DIM * id + 2] += l * norm.z;

    float3 addVel = -dot(norm, vel) * norm;

    dvel[DIM * id] += addVel.x;
    dvel[DIM * id + 1] += addVel.y;
    dvel[DIM * id + 2] += addVel.z;
}

//���Ԑϕ�
//dpos:�X�V��̈ʒu
//dcurpos:�X�V�O�̈ʒu
//dvel:���x(�X�V�������x����)
//dt:�^�C���X�e�b�v
//n:���q��
__global__
void CxIntegrate(float* dpos, float* dcurpos,float* dvel,float dt,int n,bool vel_control) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 cur_pos = make_float3(dcurpos[DIM * id], dcurpos[DIM * id + 1], dcurpos[DIM * id + 2]);
    float3 vel = (pos-cur_pos)/dt;
    
    //���Ԑϕ��̍ۂɁC��葬�x�ȉ��̗��q�ɂ��Ă͕ω��Ȃ��Ƃ��āC�Œ肵�Ă��܂�
    if (vel_control&&length(vel) < VEL_EPSILON) {
        //���x��0�ɌŒ�
        vel = make_float3(0.f);
        //�ʒu���X�V�O�̒l�ɖ߂�
        pos = cur_pos;
        dpos[DIM * id] = pos.x;
        dpos[DIM * id + 1] = pos.y;
        dpos[DIM * id + 2] = pos.z;
    }

    //if (length(vel) > 1.0e-2) printf("id %d vel x:%f,y:%f,z:%f\n", id, vel.x, vel.y, vel.z);

    dvel[DIM * id] = vel.x;
    dvel[DIM * id + 1] = vel.y;
    dvel[DIM * id + 2] = vel.z;

    dcurpos[DIM * id] = pos.x;
    dcurpos[DIM * id + 1] = pos.y;
    dcurpos[DIM * id + 2] = pos.z;
}

//�O�͌v�Z
//����d�͂��C���[�W���������x��^����
//�S�Ă̗��q�ɓ��������x��^����
//�f�o�b�N�p
__global__
void CxCalExternalForces(float* dpos,float* dvel,float* dmass,int* dfix,float3 gravity, float3 wind,float dt, int n){
int id = blockDim.x * blockIdx.x + threadIdx.x;
if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�){
if (dfix[id] == 1) return;
if (dfix[id - 1] == 1)return;

float mass = dmass[id];

dvel[DIM * id] += (gravity.x+wind.x) * dt;
dvel[DIM * id + 1] += (gravity.y+wind.y) * dt;
dvel[DIM * id + 2] += (gravity.z+wind.z) * dt;

dpos[DIM * id] += dvel[DIM * id] * dt;
dpos[DIM * id + 1] += dvel[DIM * id + 1] * dt;
dpos[DIM * id + 2] += dvel[DIM * id + 2] * dt;
}

//�ʒu�x�[�X�@�̐L�ѐ���
//�f�o�b�N�p
__global__
void CxStretchingConstraint(float* dpos, float* dmas, float* dlen, float* dkss, float* dquat, int* dfix, int n,int odd_even) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    id = id * 2 + odd_even;

    if (id >= n - 1) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    if (dfix[id + 1] == 1) return;//�e���̖��ɕӂ̐������s��

    float3 pos0 = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 pos1 = make_float3(dpos[DIM * (id + 1)], dpos[DIM * (id + 1) + 1], dpos[DIM * (id + 1) + 2]);

    float mass = dmas[id];//���݂͗��q�̏d���͑S�ē������Ƃ��Ă���
    float length = dlen[id];
    float kss = dkss[id];
    float4 quat = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);//�l������(x,y,z,w)�ɂȂ�悤�Ɏ󂯎��

    float3 e3 = make_float3(0, 0, 1);
    float3 d3 = rotVecByQuat(e3, quat);

    float w = 1.0 / mass;//�����p
    float wq = 1.0e5;//�����p
    wq = 1 / length;

    //��pos���v�Z
    float3 gamma = (pos1 - pos0) / length - d3;
    gamma /= (2*w) / length + 4.0f * wq * length;
    float ks = 1.0;
    gamma *= ks;

    float3 delta_pos0 = w * gamma;
    float3 delta_pos1 = -w * gamma;

    // calc delta_q
    float4 e3q = make_float4(e3, 0.0f);
    float4 q_e3_bar = quatProduct(quat, quatConjugate(e3q)); // calc q*e3_bar
    float4 gammaq = make_float4(gamma, 0.0f);
    float4 inter_quat = quatProduct(gammaq, q_e3_bar);
    float4 delta_quat = wq * length * quatProduct(gammaq, q_e3_bar);//2.0�ǉ�

    float4 new_quat = quat + delta_quat;//�X�V���qs
    new_quat = normalize(new_quat);

    if (dfix[id] == 0) {
        //printf("dpos %f", Length(delta_pos0));
        dpos[id * DIM] += delta_pos0.x;
        dpos[id * DIM + 1] += delta_pos0.y;
        dpos[id * DIM + 2] += delta_pos0.z;
    }

    dpos[(id + 1) * DIM] += delta_pos1.x;
    dpos[(id + 1) * DIM + 1] += delta_pos1.y;
    dpos[(id + 1) * DIM + 2] += delta_pos1.z;

    dquat[id * QUAT] = new_quat.x;
    dquat[id * QUAT + 1] = new_quat.y;
    dquat[id * QUAT + 2] = new_quat.z;
    dquat[id * QUAT + 3] = new_quat.w;
}

//�ʒu�A���x�A�����x���o��
//�f�o�b�N�p
__global__
void CxPrint3Dfloat(float* dpos, float* dvel, float* dacc, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return;

    printf("id %d\n", id);
    printf("pos x:%f,y:%f,z:%f\n", dpos[id * DIM], dpos[id * DIM + 1], dpos[id * DIM + 2]);
    printf("vel x:%f,y:%f,z:%f\n", dvel[id * DIM], dvel[id * DIM + 1], dvel[id * DIM + 2]);
    printf("acc x:%f,y:%f,z:%f\n", dacc[id * DIM], dacc[id * DIM + 1], dacc[id * DIM + 2]);
}

//�ڐ��̍X�V
//dpos:�ʒu
//dtang:�G�b�W���Ƃ̐ڐ�
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��
__global__
void CxTangUpdate(float* dpos, float* dtang, int* dfix, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return;
    if (dfix[id] == 1)return;

    dtang[DIM * id] = dpos[DIM * id] - dpos[DIM * (id - 1)];
    dtang[DIM * id + 1] = dpos[DIM * id + 1] - dpos[DIM * (id - 1) + 1];
    dtang[DIM * id + 2] = dpos[DIM * id + 2] - dpos[DIM * (id - 1) + 2];
}

//�p���x�ȂǏ�������f�o�C�X�������ɐݒ�����Ă�����̂̏����l��0�ɐݒ�
//dangvel:�p���x
//dfss:�G�b�W���Ƃɂ������(GlobalForceStep�ŋ��߂�)
//dpbf_lambda:���x����̌v�Z�ߒ��ɕK�v�ȃɂ��������m��
//n:���q��
__global__
void CxSetParametersZero(float* dangvel,float* dfss,float* dpbf_lambda, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return;
    dangvel[DIM * id] = dangvel[DIM * id + 1] = dangvel[DIM * id + 2] = 0;
    dfss[DIM * id] = dfss[DIM * id + 1] = dfss[DIM * id + 2] = 0;
    dpbf_lambda[id] = 0.f;
}

//�p�����x�̍X�V
//dangvel:�p���x
//dquat:�p��(�l����)
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//dt:�^�C���X�e�b�v
//n:���q��
__global__
void CxAngVelUpdate(float* dangvel, float* dquat,int* dfix,float dt, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n-1) return;
    if (dfix[id + 1] == 1) return;

    float4 quat = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);
    float3 avel = make_float3(dangvel[DIM * id], dangvel[DIM * id + 1], dangvel[DIM * id + 2]);

    float4 avelq = make_float4(avel, 0.0f);
    quat += 0.5f * quatProduct(quat, avelq) * dt;
    quat = normalize(quat);

    dquat[QUAT * id] = quat.x;
    dquat[QUAT * id + 1] = quat.y;
    dquat[QUAT * id + 2] = quat.z;
    dquat[QUAT * id + 3] = quat.w;
}

//�e�����x�̎��Ԑϕ�
//dangvel:�p���x
//dcurquat:�O�X�e�b�v�̎p��(�ʒu�C���O)
//dquat:���݂̎p��(�ʒu�C����)
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//dt:�^�C���X�e�b�v
//n:���q��
//vel_control:�p���x�����ȉ��Ȃ�؂�̂Ă��s�����ǂ����𔻒�
__global__
void CxAngVelIntegrate(float* dangvel,float* dcurquat, float* dquat,int* dfix,float dt, int n,bool vel_control) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n-1) return;
    if (dfix[id + 1] == 1) return;

    float4 quat = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);
    float4 cur_quat = make_float4(dcurquat[QUAT * id], dcurquat[QUAT * id + 1], dcurquat[QUAT * id + 2], dcurquat[QUAT * id + 3]);

    float4 delta_rot = quatProduct(quatConjugate(cur_quat), quat);
    float3 new_AngVel = 2.0f * make_float3(delta_rot.x, delta_rot.y, delta_rot.z) / dt;

    //�p���x�����ȉ��Ȃ�C�����Ă��Ȃ��Ƃ��ČŒ肷��
    if (vel_control&&length(new_AngVel) < ANGVEL_EPSILON) {
        new_AngVel = make_float3(0.f);
        quat = cur_quat;
        dquat[QUAT * id] = quat.x;
        dquat[QUAT * id + 1] = quat.y;
        dquat[QUAT * id + 2] = quat.z;
        dquat[QUAT * id + 3] = quat.w;
    }

    //if (length(new_AngVel) > 1.0e-2)printf("id %d angvel x:%f,y:%f,z:%f\n", id, new_AngVel.x, new_AngVel.y, new_AngVel.z);

    dangvel[DIM * id] = new_AngVel.x;
    dangvel[DIM * id + 1] = new_AngVel.y;
    dangvel[DIM * id + 2] = new_AngVel.z;

    dcurquat[QUAT * id] = quat.x;
    dcurquat[QUAT * id + 1] = quat.y;
    dcurquat[QUAT * id + 2] = quat.z;
    dcurquat[QUAT * id + 3] = quat.w;
}

//����x���ŏ��Ɍv�Z�������x�ɐݒ�
//dpos:�ʒu
//dRestDens:���q���Ƃɐݒ肷�����x
//dvol:�̐�
//dmas:����
//n:���q��
__global__
void CxRestDensSet(float* dpos,float* dRestDens,float* dvol, float*dmas,int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos0 = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //float m = params.mass;
    float a = params.aw;
    float rest_dens = params.rest_dens;

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos0 - make_float3(h));
    grid_pos1 = calcGridPos(pos0 + make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ė��x�v�Z
    float dens = 0.0f;
    for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
        for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
            for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l

                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for (uint j = startIndex; j < endIndex; ++j) {
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos1 = params.cell.dSortedPos[j];
                        float3 rij = pos0 - pos1;
                        float r = length(rij);
                        if (r <= h) {
                            // Poly6�J�[�l���Ŗ��x���v�Z (rho = �� m Wij)
                            float q = h * h - r * r;
                            // ���̗��q��rest_dens*dvol[sj] = m�ƂȂ�悤�ɐݒ肵�Ă���
                            // ���E���q�͑z�̐ςƏ������x���畡���w���E���q���������ꍇ�̉��z���ʃ�=��0*Vb�����߂Ďg�� 
                            //float m = rest_dens * dvol[sj];
                            //printf("mass %f\n", m);
                            float m = params.mass;
                            dens += m * a * q * q * q;
                        }
                    }
                }
            }
        }
    }


    // �v�Z�������x���f�o�C�X�������ɏ�������
    uint sid = params.cell.dSortedIndex[id];
    //�Œ���̖��x��ݒ�
    dRestDens[sid] = fmaxf(dens,rest_dens);//�o������̐ݒ���@
}

//�ꗥ�̊�ƂȂ閧�x�̐ݒ�
//�f�o�b�N�p
__global__
void CxRestTotalDens(float* drestdens, float dens, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    drestdens[id] = dens;
}

//�ȉ�SagFree�̏������ڐA--------------------------------------------------------------------------------------------------------------------------------
//LU������p���āA�A��1��������������
//1�Ȃ琬��,0�Ȃ玸�s
//���݁C�g�p���Ă��Ȃ�
__device__ __host__
int LUDecomp(float A[][4], int n) {
    if (n <= 0) return 0;

    for (int i = 0; i < n; i++) {
        //l_ij�̌v�Z(i>=j)
        for (int j = 0; j <= i; ++j) {
            float lu = A[i][j];
            for (int k = 0; k < j; k++) {
                lu -= A[i][k] * A[k][j];//l_ik*u_kj
            }
            A[i][j] = lu;
        }

        //u_ij�̌v�Z(i<j)
        for (int j = i + 1; j < n; ++j) {
            float lu = A[i][j];
            for (int k = 0; k < i; ++k) {
                lu -= A[i][k] * A[k][j];
            }
            A[i][j] = lu / A[i][i];
        }
    }

    return 1;
}

//A:LU�������ꂽ�s��
//b:�E�Ӄx�N�g��
//x:���ʃx�N�g��
//n:�s��̑傫��
//���݁C�g�p���Ă��Ȃ�
__device__ __host__
int LUSolver(const float A[][4], const float b[], float x[], int n) {
    if (n <= 0) return 0;

    //�O�i���
    //LY=b����Y���v�Z
    for (int i = 0; i < n; ++i) {
        float bly = b[i];
        for (int j = 0; j < i; ++j) {
            bly -= A[i][j] * x[j];
        }
        //if (A[i][i] < 1.0e-6) printf("A trace error!\n");
        x[i] = bly / A[i][i];
    }

    //��ޑ��
    //UX=Y����X���v�Z
    for (int i = n - 1; i >= 0; --i) {
        float yux = x[i];
        for (int j = i + 1; j < n; ++j) {
            yux -= A[i][j] * x[j];
        }
        x[i] = yux;
    }

    return 1;
}

//3�����x�N�g������l���������Ƃ߂�
//z��������ƂȂ���x�N�g���Ƃ���
__device__ __host__
float4 quatFromDirector(float3 d3) {
    d3 = normalize(d3);
    float3 e3 = make_float3(0.f, 0.f, 1.0f);//z������O��
    float3 w = cross(e3, d3);
    float4 q = make_float4(w, dot(e3, d3));
    q.w += length(q);
    return normalize(q);
}

//��̃x�N�g������l���������߂�
//�Е��͊�ƂȂ���
__device__ __host__
float4 quatFromTwoVectors(float3 a, float3 b) {
    float3 v0 = normalize(a);
    float3 v1 = normalize(b);
    float c = dot(v0, v1);

    float3 axis = cross(v0, v1);
    float s = sqrt((1 + c) * 2);
    float3 vec = axis / s;
    float w = s * 0.5;

    float4 quat = make_float4(vec.x,vec.y,vec.z,w);
    return normalize(quat);
}

//�Î~���C�ƂȂ�͂����߂�
//�O���[�o���X�e�b�v�ŗp����\��
//���ꂼ��̗��q������������������Ȃ��Ɩ��C������������������܂�Ȃ����߁C�K���ȕ����������Ƃ���
//���݂́C�ЂƂ܂��C�S�Ă̗��q�����ΓI�ɉ����������ɓ������̂Ƃ���(����Ȃ��Ƃ͂��蓾�Ȃ���)
__device__
float3 calcFrictionForce(float3 dir_i, float3 dir_j, float* ddens, float* drestdens, float* dvol, int id) {
    float3 pos_i = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];
    //�Î~���C�W��(�����C�W���͐Î~���C�W����0.1�{�Ƃ���)
    float mu = MU;

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos_i - make_float3(h));
    grid_pos1 = calcGridPos(pos_i + make_float3(h));

    //�ŏI�I�Ȗ��C��
    float3 x_fric = make_float3(0.f);

    for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
        for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
            for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l
                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for (uint j = startIndex; j < endIndex; ++j) {
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos_j = params.cell.dSortedPos[j];

                        //j�̗��q�̏������x
                        float restdens_j = drestdens[sj];
                        //j�̑̐�
                        float vol_j = dvol[sj];
                        //����
                        float m = restdens_j * vol_j;

                        float3 r_ij = pos_i - pos_j;
                        float r = length(r_ij);
                        if (r <= 1.0e-3) continue;
                        if (r < h) {
                            float3 v_ij = dir_i - dir_j;
                            //v_ij = make_float3(0, 1, 0);//����������Ɉ��������Ă���Ƃ���
                            v_ij = dir_i;

                            r_ij = normalize(r_ij);
                            //�Փ˖@���ɑ΂��Đ����Ȑ��������߂�
                            float3 delxn = v_ij - r_ij * dot(v_ij, r_ij);//delta x_��=v_ij-x_||

                            float q = h * h - r * r;//(h^2-||rij||^2)
                            x_fric += m / ddens[sj] * MU * delxn * params.aw * q * q * q;//aw*q^3
                        }
                    }
                }
            }
        }
    }
    return x_fric;
}

__device__ __host__
float3 CalcNormalTorque(float3 pos0, float3 pos1, float4 quat, float3 fss, float len, float mass0, float mass1, float3 gravity) {
    float3 mid = (pos0 + pos1) / 2.f;
    //�����x�N�g�������߂�
    float3 dir = rotVecByQuat(make_float3(0, 0, 1), quat);
    //0,1�̎��_�ɑ΂���d�S(�G�b�W����)����̃x�N�g��
    float3 r0 = normalize(-dir + mid) * len / 2;
    float3 r1 = normalize(dir + mid) * len / 2;

    //���͂ɂ��G�b�W�ɂ�����͂𗼕��̎��_�ƊO�ς����
    float3 tau_int0 = cross(r0, fss);
    float3 tau_int1 = cross(r1, fss);

    //���͂ɂ��g���N���o��
    float3 tau_internal = tau_int0 + tau_int1;
    //printf("id %d tau_internal x:%f,y:%f,z:%f\n",id,tau_internal.x,tau_internal.y,tau_internal.z);

    //�O��(�d��)�ɂ��g���N�����߂�
    float3 tau_ext0 = cross(r0, mass0 * gravity);
    float3 tau_ext1 = cross(r1, mass1 * gravity);

    //�O�͂ɂ��g���N���o��
    float3 tau_external = tau_ext0 + tau_ext1;
    //printf("id %d tau_external x:%f,y:%f,z:%f\n", id, tau_external.x, tau_external.y, tau_external.z);

    float3 total_torque = tau_internal + tau_external;
    //if (length(total_torque) > 1.0e-3) printf("id %d total_torque x:%f,y:%f,z:%f\n", id, total_torque.x, total_torque.y, total_torque.z);

    return total_torque;
}

//�O���[�o���t�H�[�X�X�e�b�v
//�ЂƂ܂����C�����l�����ɁA�P���ɏd�͂��狁�߂邱�ƂƂ���
//�G�b�W����ɖ�������
//dfss:�G�b�W���Ƃɂ������
//dmass:����
//last_index:�є����Ƃ̍Ō�̗��q�̃C���f�b�N�X���i�[
//gravity:�d��
//num_elastic:�����ł́C�є����Ƃɕ���v�Z���邽�߁C�є��̐���n��
__global__
void CxGlobalForceStep(float* dpos,float* dfss,float* dmass,int* dlast_ind,float3 gravity,float* ddens,float* drestdens,float* dvol,int num_elastic) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= num_elastic) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    int min;
    if (id == 0) min = 1;
    else min = dlast_ind[id - 1] + 2;//�Œ�_�ɗאڂ���G�b�W�͌v�Z�Ɋ܂܂Ȃ�
    int max = dlast_ind[id] - 1;
    for (int i = max; i > min - 1; i--) {//i=0�̎��͗�0
        float mass = dmass[i+1];

        float3 pos0 = make_float3(dpos[i * DIM], dpos[i * DIM + 1], dpos[i * DIM + 2]);
        float3 pos1 = make_float3(dpos[(i + 1) * DIM], dpos[(i + 1) * DIM + 1], dpos[(i + 1) * DIM + 2]);

        float3 dir = normalize(pos1 - pos0);

        float3 prev_fss;
        if (i == max) prev_fss = make_float3(0.f);
        else prev_fss = make_float3(dfss[(i + 1) * DIM], dfss[(i + 1) * DIM + 1], dfss[(i + 1) * DIM + 2]);

        dfss[i * DIM] = -(mass * gravity.x - prev_fss.x);
        dfss[i * DIM + 1] = -(mass * gravity.y - prev_fss.y);
        dfss[i * DIM + 2] = -(mass * gravity.z - prev_fss.z);
        
        //11/16�ǉ�
        //�G�b�W�̕����ɂ݈̂��������Ă݂�--------------------
       /* float a = -(mass * gravity.y - prev_fss.y) / dir.y;

        dfss[i * DIM] = a * dir.x;
        dfss[i * DIM + 1] = a * dir.y;
        dfss[i * DIM + 2] = a * dir.z;*/
        //----------------------------------------------------

        //float3 dir_i = make_float3(0, -1, 0);
        //float3 dir_j = make_float3(0, -1, 0);
        //����,dir_i��dir_j�ɈӖ��͂Ȃ�
        //float3 friction_force = 0.5*calcFrictionForce(dir_i, dir_j, ddens, drestdens, dvol,id);
        
        //���C���܂߂čl����
        /*dfss[i * DIM] = -(mass * gravity.x - prev_fss.x - friction_force.x);
        dfss[i * DIM + 1] = -(mass * gravity.y - prev_fss.y - friction_force.y);
        dfss[i * DIM + 2] = -(mass * gravity.z - prev_fss.z - friction_force.z);*/

        //�l��ς���ꍇ
        /*if (i == max) {
            dfss[i * DIM] *= 20;
            dfss[i * DIM + 1] *= 20;
            dfss[i * DIM + 1] *= 20;
        }*/
    }
}

//���[�J���t�H�[�X�X�e�b�v
//�O���[�o���t�H�[�X�X�e�b�v�ŋ��߂��G�b�W���Ƃ̗͂���C�ό`��h�����߂̊����p�������߂�
//dpos:�ʒu
//dlen:���
//dquat:�p��
//dcurquat:�O�X�e�b�v�̎p��(�V�~�����[�V�����J�n�O��curquat��quat�ƈ�v���邽�߁C�X�V��̒l����)
//dkss:�L�э���
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��(�G�b�W���Ƃɕ���v�Z)
__global__
void CxLocalForceStep(float* dpos, float* dlen, float* dquat,float* dcurquat, float* dkss, float* dfss, int* dfix, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n - 1) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    if (dfix[id] == 1 || dfix[id + 1] == 1) return;//�e���̖��ɕӂ̐������s���A�ŏ��̃G�b�W�͌Œ肵�Ĉ���

    float3 pos1 = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 pos2 = make_float3(dpos[DIM * (id + 1)], dpos[DIM * (id + 1) + 1], dpos[DIM * (id + 1) + 2]);

    float3 fss = make_float3(dfss[DIM * id], dfss[DIM * id + 1], dfss[DIM * id + 2]);

    float fs_Len = Length(fss);

    float l0 = dlen[id];
    float tmp_ks = dkss[id];
    float4 quat = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);//�l������(x,y,z,w)�ɂȂ�悤�Ɏ󂯎��

    float3 e3 = make_float3(0, 0, 1);
    float3 d3 = normalize(rotVecByQuat(e3, quat));

    //���ʎ��𖞂������ǂ����ɂ���āC�L�э���������������----------------
    float delta = 10.f;
    float B = dot((pos1 - pos2), fss) / (tmp_ks)+1;
    float AC = dot(pos1 - pos2, pos1 - pos2) * fs_Len * fs_Len / (tmp_ks * tmp_ks);//Length2��dot�ŕ\��
    float discrim = B * B - 4 * AC;
    //�����ŋ��߂�ꍇ
    /*while (1) {
        if (discrim > 1.0e-2) break;
        tmp_ks += delta;
    }*/

    //���ʎ��𖞂����Ȃ��ꍇ�A�莮�I��kss�����߂�
    if (discrim < 0) {
        //2������苁�߂Ă���A�{��+-sqrt(...)�ł��邪�Akss>0���A+�݂̂��l����
        tmp_ks = - abs(dot(fss, pos1 - pos2)) + 2 * fs_Len * Length(pos1 - pos2);
        printf("id %d discrim<0!!\n", id);
    }
    
    //�L�э������X�V
    dkss[id] = tmp_ks;

    //��̌��ƂȂ钷�������߂�(float�^�̂܂܂��Ɛ��x�ɉe�������邽�߁Adouble�^�ɕύX)
    double a = fs_Len * fs_Len;
    double b = -tmp_ks * (2.f * dot(fss, pos1 - pos2) + tmp_ks);
    double c = tmp_ks * tmp_ks * dot(pos1 - pos2, pos1 - pos2);//Length2��dot�ŕ\��

    //�����2�̌��
    double l1 = sqrt((-b + sqrt(abs(b * b - 4.f * a * c))) / (2.f * a));
    double l2 = sqrt((-b - sqrt(abs(b * b - 4.f * a * c))) / (2.f * a));

    //2�̌��̂����A��茻�݂̒����ɋ߂����̂�I��
    float rest_length;
    if (abs(l1 - l0) > abs(l2 - l0) && abs(l2) > 1.0e-10) rest_length = l2;
    else if (abs(l1) > 1.0e-10) rest_length = l1;
    else {
        printf("Error Occured! in LocalForceStep!");
    }

    //�p���̍X�V
    float3 new_ds = (rest_length * fss) / tmp_ks - (pos1 - pos2) / rest_length;

    //�l�������x�N�g�����狁�߂�(��̎�@�����邪�A�قƂ�Ǖς��Ȃ��Ɛ���)
    float4 new_qs = quatFromDirector(new_ds);
    //float4 new_qs = quatFromTwoVectors(e3, new_ds);

    //�p���x�N�g���̊m�F
    float3 d3_from_qs = rotVecByQuat(e3, new_qs);

    //�͂̊m�F�p(Eq.14�̏�ł́Cds3��-�Œ�`����Ă��邪�C�����+���ƍl������D)
    float3 new_Fss = (tmp_ks / rest_length) * ((pos1 - pos2) / rest_length + d3_from_qs);

    //printf("id %d old_quat x:%f,y:%f,z:%f,w:%f new_quat x:%f,y:%f,z:%f,w:%f\n", id, quat.x, quat.y, quat.z, quat.w, new_qs.x, new_qs.y, new_qs.z, new_qs.w);

    //����̍X�V
    dlen[id] = rest_length;
    //�p���̍X�V
    dquat[id * QUAT] = dcurquat[id * QUAT] = new_qs.x;
    dquat[id * QUAT + 1] = dcurquat[id * QUAT + 1] = new_qs.y;
    dquat[id * QUAT + 2] = dcurquat[id * QUAT + 2] = new_qs.z;
    dquat[id * QUAT + 3] = dcurquat[id * QUAT + 3] = new_qs.w;
}

//�L�сE����f����̃g���N�����߂�
//qs:���݁C���ڂ��Ă���G�b�W�̎p��
//pos1,pos2:�G�b�W�̗��[�̗��q�̈ʒu
//len:���
//kss:�L�э���
__device__ __host__
float4 StretchingShearTorque(float4 qs, float3 pos1, float3 pos2, float len, float kss) {
    float3 V = (pos1 - pos2) / len;

    float4 torqueSS;
    torqueSS.x = 4 * (qs.x * qs.x * qs.x + qs.x * qs.y * qs.y + qs.x * qs.z * qs.z + qs.x * qs.w * qs.w + V.z * qs.x - V.x * qs.z + V.y * qs.w);//x
    torqueSS.y = 4 * (qs.y * qs.y * qs.y + qs.y * qs.x * qs.x + qs.y * qs.z * qs.z + qs.y * qs.w * qs.w + V.z * qs.y - V.x * qs.w - V.y * qs.z);//y
    torqueSS.z = 4 * (qs.z * qs.z * qs.z + qs.z * qs.x * qs.x + qs.z * qs.y * qs.y + qs.z * qs.w * qs.w - V.z * qs.z - V.x * qs.x - V.y * qs.y);//z
    torqueSS.w = 4 * (qs.w * qs.w * qs.w + qs.w * qs.x * qs.x + qs.w * qs.y * qs.y + qs.w * qs.z * qs.z - V.z * qs.w - V.x * qs.y + V.y * qs.x);//w
    torqueSS = 1.f / 2.f * kss * torqueSS;

   /* float a = qs.x;
    float b = qs.y;
    float c = qs.z;
    float d = qs.w;

    torqueSS.x = 2 * (V.x - 2 * a * c - 2 * b * d) * (-2 * c) + 2 * (V.y + 2 * a * d - 2 * b * c) * (2 * d) + 2 * (V.z - d * d + a * a + b * b + c * c) * 2 * a;
    torqueSS.y = 2 * (V.x - 2 * a * c - 2 * b * d) * (-2 * d) + 2 * (V.y + 2 * a * d - 2 * b * c) * (-2 * c) + 2 * (V.z - d * d + a * a + b * b + c * c) * 2 * b;
    torqueSS.z = 2 * (V.x - 2 * a * c - 2 * b * d) * (-2 * a) + 2 * (V.y + 2 * a * d - 2 * b * c) * (-2 * b) + 2 * (V.z - d * d + a * a + b * b + c * c) * 2 * c;
    torqueSS.w = 2 * (V.x - 2 * a * c - 2 * b * d) * (-2 * b) + 2 * (V.y + 2 * a * d - 2 * b * c) * (2 * a) + 2 * (V.z - d * d + a * a + b * b + c * c) * (-2 * d);

    torqueSS *= 1.f / 2.f * kss;*/

    return torqueSS;
}

//�Ȃ��E�˂��ꐧ��̃g���N�����߂�(���ݗ��p����)
__device__ __host__
float4 BendTwistTorque(float4 q1, float4 q2, float4 Darboux, float kbt) {
    float4 torqueBT;
    float sum = q2.x * q2.x + q2.y * q2.y + q2.z * q2.z + q2.w * q2.w;
    torqueBT.x = sum * q1.x - q2.x * Darboux.w + q2.y * Darboux.z - q2.z * Darboux.y + q2.w * Darboux.x;
    torqueBT.y = sum * q1.y - q2.x * Darboux.z - q2.y * Darboux.w + q2.z * Darboux.x + q2.w * Darboux.y;
    torqueBT.z = sum * q1.z + q2.x * Darboux.y - q2.y * Darboux.x - q2.z * Darboux.w + q2.w * Darboux.z;
    torqueBT.w = sum * q1.w - q2.x * Darboux.x - q2.y * Darboux.y - q2.z * Darboux.z - q2.w * Darboux.w;

    return kbt * torqueBT;
}

//�g���N���狁�߂�ꍇ�ɗ��p(���ݗ��p����)
__device__ __host__
float4 SolveTorqueSolver(float4 q1, float4 q2, float4 torqueSS, float4 torqueBT, float kbt) {
    float4 rightForm = -(torqueSS + torqueBT);
    rightForm = rightForm / kbt;

    float4 New_Darboux;

    float sum = q2.x * q2.x + q2.y * q2.y + q2.z * q2.z + q2.w * q2.w;
    float b[4];
    b[0] = rightForm.x - sum * q1.x;
    b[1] = rightForm.y - sum * q1.y;
    b[2] = rightForm.z - sum * q1.z;
    b[3] = rightForm.w - sum * q1.w;

    float A[4][4];
    A[0][0] = q2.w;
    A[0][1] = -q2.z;
    A[0][2] = q2.y;
    A[0][3] = -q2.x;

    A[1][0] = q2.z;
    A[1][1] = q2.w;
    A[1][2] = -q2.x;
    A[1][3] = -q2.y;

    A[2][0] = -q2.y;
    A[2][1] = q2.x;
    A[2][2] = q2.w;
    A[2][3] = -q2.z;

    A[3][0] = -q2.x;
    A[3][1] = -q2.y;
    A[3][2] = -q2.z;
    A[3][3] = -q2.w;

    float x[4];
    int n1=LUDecomp(A, 4);
    //if (n1 == 0) printf("LUDecomp failure!!\n");
    int n2=LUSolver(A, b, x, 4);
    //if (n2 == 0)printf("LUSolver failure!!\n");

    New_Darboux = make_float4(x[0], x[1], x[2], x[3]);
    //printf("%d:New_Darboux x:%f,y:%f,z:%f,w:%\nf", New_Darboux.x, New_Darboux.y, New_Darboux.z, New_Darboux.w);
    return New_Darboux;
}

//�l�����̋t����Ԃ�
__device__ __host__
float4 QuatInverse(float4 quat) {
    return quatConjugate(quat) / dot(quat, quat);
}

//����̕����Q�l�ɐ��`�V�X�e���I�ɉ������C���ۂɂ́A��ԉ��̃G�b�W���珇�Ԃɉ����Ă���
//��ڂ̃G�b�W�͊��S�ɌŒ肷��̂ŁA��ڂ̃G�b�W�܂ł��l����
//�אڃG�b�W�̍X�V��̒l���g���K�v������̂ŁA�є��P�ʂł̕��񉻂ɂȂ�
//dpos:�ʒu
//dquat:�p��
//domega:��_���{�[�x�N�g��
//dlen:���
//dkss:�L�э���
//dkbt:�Ȃ�����
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//last_index:�є����Ƃ̍Ō�̗��q�̃C���f�b�N�X���i�[
//num_elastic:�����ł́C�є����Ƃɕ���v�Z���邽�߁C�є��̐���n��
__global__
void CxGlobalTorqueStep(float* dpos, float* dquat, float* domega, float* dlen, float* dkss, float* dkbt, int* dfix, int* dlast_index, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    int last_edge = dlast_index[id] - 1;//�Ō�̗��q-1���Ō�̃G�b�W
    float min;
    if (id == 0) min = 0;
    else min = dlast_index[id - 1]+1;//�ŏ��̗��q

    //��ԉ��̃G�b�W(i==last_edge)�̏����͑��ƈႢ�A��݂̂̊�_���{�[�x�N�g�����ւ��B��x����Ȃ̂ŁA���[�v���g�킸�ɏ������C�����͈ȍ~�̃��[�v�Ƃقړ���
    float3 init_pos1 = make_float3(dpos[last_edge * DIM], dpos[last_edge * DIM + 1], dpos[last_edge * DIM + 2]);
    float3 init_pos2 = make_float3(dpos[(last_edge + 1) * DIM], dpos[(last_edge + 1) * DIM + 1], dpos[(last_edge + 1) * DIM + 2]);

    float4 init_quat0= make_float4(dquat[(last_edge - 1) * QUAT], dquat[(last_edge - 1) * QUAT + 1], dquat[(last_edge - 1) * QUAT + 2], dquat[(last_edge - 1) * QUAT + 3]);
    float4 init_quat1= make_float4(dquat[last_edge * QUAT], dquat[last_edge * QUAT + 1], dquat[last_edge * QUAT + 2], dquat[last_edge * QUAT + 3]);

    float init_l0 = dlen[last_edge];
    float init_kss = dkss[last_edge];
    float init_kbt = dkbt[last_edge];

    float init_K = init_kbt;
    float4 init_kq_inv = QuatInverse(init_K * init_quat1);

    float4 init_torqueSS = StretchingShearTorque(init_quat1, init_pos1, init_pos2, init_l0, init_kss);
    init_torqueSS = quatProduct(init_torqueSS, init_kq_inv);//�K���ɒ萔�������Ă݂�

    float4 init_Cur_Omega_Prev = quatProduct(quatConjugate(init_quat0), init_quat1);
    float4 init_Rest_Omega_Prev = init_Cur_Omega_Prev - init_torqueSS;
    
    //printf("id %d init_torqueSS x:%f,y:%f,z:%f,w:%f\n", id, init_torqueSS.x, init_torqueSS.y, init_torqueSS.z, init_torqueSS.w);
    //printf("id %d init_Cur_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, init_Cur_Omega_Prev.x, init_Cur_Omega_Prev.y, init_Cur_Omega_Prev.z, init_Cur_Omega_Prev.w);
    //printf("id %d init_Rest_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, init_Rest_Omega_Prev.x, init_Rest_Omega_Prev.y, init_Rest_Omega_Prev.z, init_Rest_Omega_Prev.w);

    //init_Rest_Omega_Prev = quatConjugate(init_Rest_Omega_Prev);
    //�_���{�[�x�N�g���̕������Ȃ��˂��ꐧ��̂悤�ɋ��߂�K�v�����邪�A�ق�1�ł���Ɛ����ł��邽�߁A1�ň���(s*Omega�����̂܂܃_���{�[�x�N�g���̔z��ɓ����)
    domega[(last_edge - 1) * QUAT] = init_Rest_Omega_Prev.x;
    domega[(last_edge - 1) * QUAT + 1] = init_Rest_Omega_Prev.y;
    domega[(last_edge - 1) * QUAT + 2] = init_Rest_Omega_Prev.z;
    domega[(last_edge - 1) * QUAT + 3] = init_Rest_Omega_Prev.w;
    //-----------------------------------------------------------------------------------------------------------------------------

    //���̃G�b�W�̏���
    for (int i = last_edge-1; i > min; i--) {//i�̂ЂƂO�̊�_���{�[�x�N�g�������߂�
        float3 pos1 = make_float3(dpos[i * DIM], dpos[i * DIM + 1], dpos[i * DIM + 2]);
        float3 pos2 = make_float3(dpos[(i + 1) * DIM], dpos[(i + 1) * DIM + 1], dpos[(i + 1) * DIM + 2]);

        float4 quat0 = make_float4(dquat[(i - 1) * QUAT], dquat[(i - 1) * QUAT + 1], dquat[(i - 1) * QUAT + 2], dquat[(i - 1) * QUAT + 3]);
        float4 quat1 = make_float4(dquat[i * QUAT], dquat[i * QUAT + 1], dquat[i * QUAT + 2], dquat[i * QUAT + 3]);
        float4 quat2 = make_float4(dquat[(i + 1) * QUAT], dquat[(i + 1) * QUAT + 1], dquat[(i + 1) * QUAT + 2], dquat[(i + 1) * QUAT + 3]);

        float l0 = dlen[i];
        float kss = dkss[i];
        float kbt = dkbt[i];

        //�L�сE����f����̃g���N������Ŋ���
        float K = kbt;
        float4 kq_inv = QuatInverse(K * quat1);

        //�L�сE����f����̃g���N�����߂�
        float4 torqueSS = StretchingShearTorque(quat1, pos1, pos2, l0, kss);
        torqueSS = quatProduct(torqueSS, kq_inv);//100�{

        float4 Cur_Omega_Prev = quatProduct(quatConjugate(quat0), quat1);//���݂̃G�b�W�ƂЂƂO�̃G�b�W�̃_���{�[�x�N�g��
        float4 Cur_Omega_Next = quatConjugate(quatProduct(quatConjugate(quat1), quat2));//���݂̃G�b�W�ƈ�ۂ̃G�b�W�̃_���{�[�x�N�g��
        //�����@���ŋ��߂�(�v�Z���ʂ͕ς��Ȃ���)
        //Cur_Omega_Next = quatProduct(quatConjugate(quat2), quat1);
        
        float4 Rest_Omega_Next = make_float4(domega[i * QUAT], domega[i * QUAT + 1], domega[i * QUAT + 2], domega[i * QUAT + 3]);
        //�ŏI�I�ɋ��߂��_���{�[�x�N�g��
        float4 Rest_Omega_Prev = Cur_Omega_Next + Cur_Omega_Prev - Rest_Omega_Next - torqueSS;//�ЂƂO�̃G�b�W�Ƃ̊Ԃ̊�_���{�[�x�N�g��(Appendix������torqueSS��+�ɕύX)
        Rest_Omega_Prev = quatConjugate(Rest_Omega_Prev);
        //���ʂ��o��
        /*if (i == min + 1) {
            printf("id %d torqueSS x:%f,y:%f,z:%f,w:%f\n", id, torqueSS.x, torqueSS.y, torqueSS.z, torqueSS.w);
            printf("id %d Cur_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, Cur_Omega_Prev.x, Cur_Omega_Prev.y, Cur_Omega_Prev.z, Cur_Omega_Prev.w);
            printf("id %d Rest_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, Rest_Omega_Prev.x, Rest_Omega_Prev.y, Rest_Omega_Prev.z, Rest_Omega_Prev.w);
        }*/

        //�_���{�[�x�N�g���̕������Ȃ��˂��ꐧ��̂悤�ɋ��߂�K�v�����邪�A�ق�1�ł���Ɛ����ł��邽�߁A1�ň���(s*Omega�����̂܂܃_���{�[�x�N�g���̔z��ɓ����)
        domega[(i - 1) * QUAT] = Rest_Omega_Prev.x;
        domega[(i - 1) * QUAT + 1] = Rest_Omega_Prev.y;
        domega[(i - 1) * QUAT + 2] = Rest_Omega_Prev.z;
        domega[(i - 1) * QUAT + 3] = Rest_Omega_Prev.w;
    }
}

//���܂łƂ͋t�ɏォ������悤�ɂ���
//
__global__
void CxGlobalTorqueStep_Upstair(float* dpos, float* dquat, float* domega, float* dlen, float* dkss, float* dkbt, int* dfix, int* dlast_index, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    int last_edge = dlast_index[id] - 1;//�Ō�̗��q-1���Ō�̃G�b�W
    int min;
    if (id == 0) min = 1;
    else min = dlast_index[id - 1] + 2;//�ŏ��̗��q

    //��ԉ��̃G�b�W(i==last_edge)�̏����͑��ƈႢ�A��݂̂̊�_���{�[�x�N�g�����ւ��B��x����Ȃ̂ŁA���[�v���g�킸�ɏ������C�����͈ȍ~�̃��[�v�Ƃقړ���
    float3 init_pos1 = make_float3(dpos[min * DIM], dpos[min * DIM + 1], dpos[min * DIM + 2]);
    float3 init_pos2 = make_float3(dpos[(min + 1) * DIM], dpos[(min + 1) * DIM + 1], dpos[(min + 1) * DIM + 2]);

    float4 init_quat0 = make_float4(dquat[(min - 1) * QUAT], dquat[(min - 1) * QUAT + 1], dquat[(min - 1) * QUAT + 2], dquat[(min - 1) * QUAT + 3]);
    float4 init_quat1 = make_float4(dquat[min * QUAT], dquat[min * QUAT + 1], dquat[min * QUAT + 2], dquat[min * QUAT + 3]);

    float init_l0 = dlen[min];
    float init_kss = dkss[min];
    float init_kbt = dkbt[min];

    float init_K = init_kbt;
    float4 init_kq_inv = QuatInverse(init_K * init_quat1);

    float4 init_torqueSS = StretchingShearTorque(init_quat1, init_pos1, init_pos2, init_l0, init_kss);
    init_torqueSS = quatProduct(init_torqueSS, init_kq_inv);//�K���ɒ萔�������Ă݂�

    float4 init_Cur_Omega_Prev = quatProduct(quatConjugate(init_quat0), init_quat1);
    float4 init_Rest_Omega_Prev = init_Cur_Omega_Prev - init_torqueSS;

    //printf("id %d init_torqueSS x:%f,y:%f,z:%f,w:%f\n", id, init_torqueSS.x, init_torqueSS.y, init_torqueSS.z, init_torqueSS.w);
    //printf("id %d init_Cur_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, init_Cur_Omega_Prev.x, init_Cur_Omega_Prev.y, init_Cur_Omega_Prev.z, init_Cur_Omega_Prev.w);
    //printf("id %d init_Rest_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, init_Rest_Omega_Prev.x, init_Rest_Omega_Prev.y, init_Rest_Omega_Prev.z, init_Rest_Omega_Prev.w);

    //init_Rest_Omega_Prev = quatConjugate(init_Rest_Omega_Prev);
    //�_���{�[�x�N�g���̕������Ȃ��˂��ꐧ��̂悤�ɋ��߂�K�v�����邪�A�ق�1�ł���Ɛ����ł��邽�߁A1�ň���(s*Omega�����̂܂܃_���{�[�x�N�g���̔z��ɓ����)
    domega[min * QUAT] = init_Rest_Omega_Prev.x;
    domega[min * QUAT + 1] = init_Rest_Omega_Prev.y;
    domega[min * QUAT + 2] = init_Rest_Omega_Prev.z;
    domega[min * QUAT + 3] = init_Rest_Omega_Prev.w;
    //-----------------------------------------------------------------------------------------------------------------------------

    //���̃G�b�W�̏���
    for (int i = min+1; i < last_edge-1; i++) {//i�̂ЂƂO�̊�_���{�[�x�N�g�������߂�
        float3 pos1 = make_float3(dpos[i * DIM], dpos[i * DIM + 1], dpos[i * DIM + 2]);
        float3 pos2 = make_float3(dpos[(i + 1) * DIM], dpos[(i + 1) * DIM + 1], dpos[(i + 1) * DIM + 2]);

        float4 quat0 = make_float4(dquat[(i - 1) * QUAT], dquat[(i - 1) * QUAT + 1], dquat[(i - 1) * QUAT + 2], dquat[(i - 1) * QUAT + 3]);
        float4 quat1 = make_float4(dquat[i * QUAT], dquat[i * QUAT + 1], dquat[i * QUAT + 2], dquat[i * QUAT + 3]);
        float4 quat2 = make_float4(dquat[(i + 1) * QUAT], dquat[(i + 1) * QUAT + 1], dquat[(i + 1) * QUAT + 2], dquat[(i + 1) * QUAT + 3]);

        float l0 = dlen[i];
        float kss = dkss[i];
        float kbt = dkbt[i];

        //�L�сE����f����̃g���N������Ŋ���
        float K = kbt;
        float4 kq_inv = QuatInverse(K * quat1);

        //�L�сE����f����̃g���N�����߂�
        float4 torqueSS = StretchingShearTorque(quat1, pos1, pos2, l0, kss);
        torqueSS = quatProduct(torqueSS, kq_inv);

        float4 Cur_Omega_Prev = quatProduct(quatConjugate(quat0), quat1);//���݂̃G�b�W�ƂЂƂO�̃G�b�W�̃_���{�[�x�N�g��
        float4 Cur_Omega_Next = quatConjugate(quatProduct(quatConjugate(quat1), quat2));//���݂̃G�b�W�ƈ�ۂ̃G�b�W�̃_���{�[�x�N�g��
        //�����@���ŋ��߂�(�v�Z���ʂ͕ς��Ȃ���)
        //Cur_Omega_Next = quatProduct(quatConjugate(quat2), quat1);

        float4 Rest_Omega_Next = make_float4(domega[(i - 1) * QUAT], domega[(i - 1) * QUAT + 1], domega[(i - 1) * QUAT + 2], domega[(i - 1) * QUAT + 3]);
        //�ŏI�I�ɋ��߂��_���{�[�x�N�g��
        float4 Rest_Omega_Prev = Cur_Omega_Next + Cur_Omega_Prev - Rest_Omega_Next - torqueSS;//�ЂƂO�̃G�b�W�Ƃ̊Ԃ̊�_���{�[�x�N�g��(Appendix������torqueSS��+�ɕύX)
        Rest_Omega_Prev = quatConjugate(Rest_Omega_Prev);
        //���ʂ��o��
        /*if (i == min + 1) {
            printf("id %d torqueSS x:%f,y:%f,z:%f,w:%f\n", id, torqueSS.x, torqueSS.y, torqueSS.z, torqueSS.w);
            printf("id %d Cur_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, Cur_Omega_Prev.x, Cur_Omega_Prev.y, Cur_Omega_Prev.z, Cur_Omega_Prev.w);
            printf("id %d Rest_Omega_Prev x:%f,y:%f,z:%f,w:%f\n", id, Rest_Omega_Prev.x, Rest_Omega_Prev.y, Rest_Omega_Prev.z, Rest_Omega_Prev.w);
        }*/

        //�_���{�[�x�N�g���̕������Ȃ��˂��ꐧ��̂悤�ɋ��߂�K�v�����邪�A�ق�1�ł���Ɛ����ł��邽�߁A1�ň���(s*Omega�����̂܂܃_���{�[�x�N�g���̔z��ɓ����)
        domega[i * QUAT] = Rest_Omega_Prev.x;
        domega[i * QUAT + 1] = Rest_Omega_Prev.y;
        domega[i * QUAT + 2] = Rest_Omega_Prev.z;
        domega[i * QUAT + 3] = Rest_Omega_Prev.w;
    }
}


//LocalTorqueStep�������̂Ɏg��
//cur_omega:���݂̓�̎p�����狁�߂�_���{�[�x�N�g��
//rest_omega:GlobalTorqueStep�ŋ��߂����K���O�̊�_���{�[�x�N�g��
//bendK:�Ȃ�����
//K_min:LocalTorqueStep�ł̒����p�����[�^
__device__ __host__
float4 solveInverseRot(float4 cur_omega, float4 rest_omega, float& bendK,float K_min) {
    const float SAFETY_FACTOR = min(abs(cur_omega.w), K_min);//0.00002f,0.002f,�ŏI�I�ɂ�0.005f
    //const float SAFETY_FACTOR = min(length(make_float3(cur_omega.x, cur_omega.y, cur_omega.z)), 0.2f);
    
    rest_omega -= dot(rest_omega, cur_omega) * cur_omega;

    //printf("omega_orth x:%f,y:%f,z:%f,w:%f\n", rest_omega.x, rest_omega.y, rest_omega.z, rest_omega.w);
    //printf("omega_orth length %f\n", length(rest_omega));

    float4 Omega = -rest_omega / bendK;

    //if (SAFETY_FACTOR > 0.2 + 1.0e-3 || SAFETY_FACTOR < 0.2 - 1.0e-3)printf("SAFETY_FACTOR %f\n", SAFETY_FACTOR);

    if (dot(Omega,Omega) > SAFETY_FACTOR * SAFETY_FACTOR) {//Length2��dot�ɒu������
        bendK = Length(rest_omega) / SAFETY_FACTOR;
        Omega = -rest_omega / bendK;
    }

    float4 d = sqrt(1 - dot(Omega, Omega)) * cur_omega;//Length2��dot�ɒu������
    Omega += d;

    return Omega;
}

//���[�J���g���N�X�e�b�v
//��_���{�[�x�N�g����K�؂Ȍ`�Ő��K������
//dquat:�p��
//domega:��_���{�[�x�N�g��
//deln:���
//dkbt:�Ȃ�����
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//K_min:LocalTorqueStep�ł̒����p�����[�^
//n:���q��(��_���{�[�x�N�g�����Ƃɕ���v�Z)
__global__
void CxLocalTorqueStep(float* dquat, float* domega, float* dlen,float* dkbt, int* dfix,float K_min,int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n - 2) return; // ���q���𒴂���X���b�hID�̃`�F�b�N
    if (dfix[id + 1] == 1 || dfix[id + 2] == 1) return;//��_���{�[�x�N�g�����Ȃ������̓X�L�b�v

    float4 quat1 = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);
    float4 quat2 = make_float4(dquat[QUAT * (id + 1)], dquat[QUAT * (id + 1) + 1], dquat[QUAT * (id + 1) + 2], dquat[QUAT * (id + 1) + 3]);
    float4 cur_omega = quatProduct(quatConjugate(quat1), quat2);//���݂̃_���{�[�x�N�g��

    float4 rest_omega = make_float4(domega[QUAT * id], domega[QUAT * id + 1], domega[QUAT * id + 2], domega[QUAT * id + 3]);

    float length = dlen[id];
    float tmp_kbt = dkbt[id];

    float4 last_omega = solveInverseRot(cur_omega, rest_omega, tmp_kbt, K_min);

    //�Ȃ��˂��ꐧ��̍����̍X�V
    dkbt[id] = tmp_kbt;
    //��_���{�[�x�N�g���̍X�V
    domega[QUAT * id] = last_omega.x;
    domega[QUAT * id + 1] = last_omega.y;
    domega[QUAT * id + 2] = last_omega.z;
    domega[QUAT * id + 3] = last_omega.w;
}

//���x����
//���x����ɗp����lambda�̌v�Z
//ddens:���݂̖��x
//drestdens:����x
//dpbf_lambda:���x����ɗp�����
//dvol:�̐�
//n:���q��
__global__
void CxPbfLambda(float* ddens,float* drestdens,float* dpbf_lambda,float* dvol,float* dmas,int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos_i = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //float m = params.mass;
    float a = params.aw;
    float rest_dens = params.rest_dens;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];
    //�C�V��SPH�ǉ�
    rest_dens = drestdens[sid];
    float dens = ddens[sid];

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos_i - make_float3(h));
    grid_pos1 = calcGridPos(pos_i + make_float3(h));
    //---------------------------------------------

    //���x���琧��̌v�Z�ɗp����ɂ����߂�
    float C = dens / rest_dens - 1.0f;
    if (C > 0.f) {
        float lambda_denom_i = 0.f;
        float3 grad_i_C = make_float3(0.f);
        for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
            for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
                for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                    int3 ngrid = make_int3(x, y, z);
                    uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l

                    // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                    uint startIndex = params.cell.dCellStart[ghash];
                    if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                        // �Z�����̃p�[�e�B�N���Ŕ���
                        uint endIndex = params.cell.dCellEnd[ghash];
                        for (uint j = startIndex; j < endIndex; ++j) {
                            uint sj = params.cell.dSortedIndex[j];
                            float3 pos_j = params.cell.dSortedPos[j];
                            float rest_dens_j = drestdens[sj];

                            //�L�����a���ɑ��݂��邩���m�F
                            float3 r_ij = pos_i - pos_j;
                            float r = length(r_ij);

                            if (r <= 1.0e-3f) continue;
                            if (r < h) {
                                float q = h - r;
                                //float m = rest_dens * dvol[sj];//�������x�Ƒ̐ς���v�Z
                                float m = dmas[sj];
                                //Grad_J_C�����߂�
                                float3 grad_j_C = -m / rest_dens_j * (-params.ag * q * q * r_ij / r);//params.ag�ɃJ�[�l���̌��z�萔(spiky�J�[�l��)���i�[����Ă���.m/\rho_rest*W(�J�[�l��)
                                lambda_denom_i += dot(grad_j_C, grad_j_C);
                                grad_i_C += grad_j_C;
                            }
                        }
                    }
                }
            }
        }
        lambda_denom_i += dot(grad_i_C, grad_i_C);
        //�ɂɒl���i�[(float�^)
        dpbf_lambda[sid] = -C / (lambda_denom_i + 1.0e-6f);//�Â�1.0e-6f�Œ�`
    }

    else {
        dpbf_lambda[sid] = 0.f;
    }
}

//���x����ɂ��ʒu�C��
//dpos:�ʒu
//drestdens:����x
//dpbf_lambda:���x����ɗp�����
//dvol:�̐�
//n:���q��
__global__
void CxPbfConstraint(float*dpos,float* drestdens,float* dpbf_lambda,float* dvol,float* dmas,int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos_i = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];
    //pbf�̌v�Z�ɕK�v�ȃɂ������Ă���
    float pbf_lambda_i = dpbf_lambda[sid];
    //�������x
    float restdens_i = drestdens[id];

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos_i - make_float3(h));
    grid_pos1 = calcGridPos(pos_i + make_float3(h));

    // ���͂̃O���b�h�Z�����܂߂ċߖT�T�����Ė��x�v�Z
    float3 delta_pos_i = make_float3(0.f);
    for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
        for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
            for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l
                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for (uint j = startIndex; j < endIndex; ++j) {
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos_j = params.cell.dSortedPos[j];
                        float restdens_j = drestdens[sj];
                        float pbf_lambda_j = dpbf_lambda[sj];

                        float3 r_ij = pos_i - pos_j;
                        float r = length(r_ij);
                        if (r <= 1.0e-3) continue;
                        if (r < h) {
                            float q;
                            q = h - r;
                            float m = restdens_j * dvol[sj];//�������x�Ƒ̐ς��玿�ʂ��v�Z
                            //printf("pbf mass %f\n", m);
                            m = dmas[sid];
                            delta_pos_i += m / restdens_i * (pbf_lambda_i + pbf_lambda_j) * (params.ag * q * q * r_ij / r);
                        }
                    }
                }
            }
        }
    }

    //���ʂ̊i�[
    dpos[DIM * sid] += delta_pos_i.x;
    dpos[DIM * sid + 1] += delta_pos_i.y;
    dpos[DIM * sid + 2] += delta_pos_i.z;
}

//SPH��PBF�ŉ����ꍇ�̈��͌v�Z���������ꍇ
//dacc:SPH�ł̈ʒu���X�V����ۂɗp��������x
//datt:���q����(0�ŗ���,1�ŋ��E)
//power:���Ȃǂ̗�
//n:���q��
__global__
void CxPbfExternalForces(float* dacc, int* datt, float3 power, bool m_wind_flag, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return;

    uint sid = params.cell.dSortedIndex[id];
    if (datt[sid] != 0) {  // ���E���q�̏ꍇ�͗��q�ɂ������=0
        float3 v0 = make_float3(0.0f);
        dacc[DIM * sid + 0] = v0.x;  dacc[DIM * sid + 1] = v0.y; dacc[DIM * sid + 2] = v0.z;
        return;
    }

    float3 acc = make_float3(dacc[DIM * id], dacc[DIM * id + 1], dacc[DIM * id + 2]);
    acc = params.gravity;

    if (m_wind_flag) {
        acc += power;
    }

    dacc[DIM * id] = acc.x;
    dacc[DIM * id + 1] = acc.y;
    dacc[DIM * id + 2] = acc.z;
}

//���C����̎���
//���C����͂Ƃ肠�����C���x�X�V�̍ۂɁC��x�̂ݍs���悤�ɐݒ肷��D
//��������ꍇ�ɂ�XPBD�ɂ��Ȃ��ƁC�����Ɉˑ����ďC���ʂ��ω�����ƍl�����邪�C���C����͐������C�����킯�ł͂Ȃ��D
//���ӗ��q���厖�Ȃ̂ŁCSort���ꂽ�ʒu�𗘗p����
//dpos:�ʒu
//dcurpos:�ʒu�C���O�̈ʒu
//drestdens:����x
//dvol:�̐�(���ʂ����z���ʂ����_0*V�Œ�`)
//ddens:���݂̖��x
//dfix:dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��
__global__
void CxFrictionConstraint(float* dpos, float* dcurpos,float* drestdens,float* dvol,float*ddens, int* dfix, int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos_i = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];

    if (dfix[sid] == 1) return;//�Œ�_�ł���΁C�X�L�b�v

    //�O�X�e�b�v�̈ʒu
    float3 cur_pos_i = make_float3(dcurpos[sid * DIM], dcurpos[sid * DIM + 1], dcurpos[sid * DIM + 2]);
    //�ʒu�C���Ȃǂɂ��ړ���
    float3 v_i = pos_i - cur_pos_i;
    //�Î~���C�W��
    float mu = MU;

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos_i - make_float3(h));
    grid_pos1 = calcGridPos(pos_i + make_float3(h));

    //�ŏI�I�Ȗ��C��
    float3 x_fric = make_float3(0.f);

    for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
        for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
            for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l
                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for (uint j = startIndex; j < endIndex; ++j) {
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos_j = params.cell.dSortedPos[j];

                        //�O�X�e�b�v�̈ʒu
                        float3 cur_pos_j = make_float3(dcurpos[sj * DIM], dcurpos[sj * DIM + 1], dcurpos[sj * DIM + 2]);
                        //j�̗��q�̏������x
                        float restdens_j = drestdens[sj];
                        //j�̑̐�
                        float vol_j = dvol[sj];
                        //����
                        float m = restdens_j * vol_j;

                        float3 r_ij = pos_i - pos_j;
                        float r = length(r_ij);
                        if (r <= 1.0e-3) continue;
                        if (r < h) {
                            //�����ɖ��C���������
                            float3 v_j = pos_j - cur_pos_j;

                            float3 v_ij = v_i - v_j;

                            r_ij = normalize(r_ij);
                            //�Փ˖@���ɑ΂��Đ����Ȑ��������߂�
                            float3 delxn = v_ij - r_ij * dot(v_ij, r_ij);//delta x_��=v_ij-x_||

                            float q = h * h - r * r;//(h^2-||rij||^2)
                            x_fric += m / ddens[sj] * MU * delxn * params.aw * q * q * q;//aw*q^3
                        }
                    }
                }
            }
        }
    }

    //i�̈ړ��ʂɂ�����x_friction�����̐��������߂�
    float3 norm_x_fric = normalize(x_fric);
    float3 dir_i_fric = norm_x_fric * dot(v_i, norm_x_fric);

    //printf("id %d friction delta x:%f,y:%f,z:%f\n",id, x_fric.x, x_fric.y, x_fric.z);

    float3 delta_x;
    if (length(dir_i_fric) <= length(x_fric)) {//�Î~���C�͂Ƃ��Ĉ����p�^�[��
        delta_x = -dir_i_fric;
    }
    else {//�����C�͂Ƃ��Ĉ����p�^�[��
        //�����炭��肠��
        delta_x = -x_fric * min(MU / length(x_fric), 1.0f);
        //delta_x = -dir_i_fric * 0.1;
        //delta_x = x_fric * 0.1;
        //delta_x = make_float3(0.f);
    }

    //�ʒu�C���ɂ��X�V
    dpos[DIM * sid] += delta_x.x;
    dpos[DIM * sid + 1] += delta_x.y;
    dpos[DIM * sid + 2] += delta_x.z;
}

//���C����̎���
//�p����L�сE����f����̈ʒu�C���ʂɊ�Â��āC��x�ɂ���ĕω������Ă݂�
__global__
void CxFrictionConstraint_withQuat(float* dpos, float* dcurpos, float* drestdens, float* dvol, float* ddens,float* dquat,float* dlen, int* dfix, int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)
    //if (dfix[id] == 1) return;//�Œ�_�ł���΁C�X�L�b�v

    float3 pos_i = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];
    //�O�X�e�b�v�̈ʒu
    float3 cur_pos_i = make_float3(dcurpos[sid * DIM], dcurpos[sid * DIM + 1], dcurpos[sid * DIM + 2]);

    if (dfix[sid] == 1) return;
    //�ЂƂO��quat----------------------
    float4 quat1 = make_float4(dquat[QUAT * (sid - 1)], dquat[QUAT * (sid - 1) + 1], dquat[QUAT * (sid - 1) + 2], dquat[QUAT * (sid - 1) + 3]);
    //����quat
    float4 quat2 = make_float4(dquat[QUAT * sid], dquat[QUAT * sid + 1], dquat[QUAT * sid + 2], dquat[QUAT * sid + 3]);
    float len = dlen[sid];
    //-----------------------------------

    //�ʒu�C���Ȃǂɂ��ړ���
    float3 v_i = pos_i - cur_pos_i;
    //�Î~���C�W��(�����C�W���͐Î~���C�W����0.1�{�Ƃ���)
    float mu = MU;

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos_i - make_float3(h));
    grid_pos1 = calcGridPos(pos_i + make_float3(h));

    //�ŏI�I�Ȗ��C��
    float3 x_fric = make_float3(0.f);

    for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
        for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
            for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l
                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for (uint j = startIndex; j < endIndex; ++j) {
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos_j = params.cell.dSortedPos[j];

                        //�O�X�e�b�v�̈ʒu
                        float3 cur_pos_j = make_float3(dcurpos[sj * DIM], dcurpos[sj * DIM + 1], dcurpos[sj * DIM + 2]);
                        //j�̗��q�̏������x
                        float restdens_j = drestdens[sj];
                        //j�̑̐�
                        float vol_j = dvol[sj];
                        //����
                        float m = restdens_j * vol_j;

                        float3 r_ij = pos_i - pos_j;
                        float r = length(r_ij);
                        if (r <= 1.0e-3) continue;
                        if (r < h) {
                            //�����ɖ��C���������
                            float3 v_j = pos_j - cur_pos_j;

                            float3 v_ij = v_i - v_j;

                            r_ij = normalize(r_ij);
                            //�Փ˖@���ɑ΂��Đ����Ȑ��������߂�
                            float3 delxn = v_ij - r_ij * dot(v_ij, r_ij);//delta x_��=v_ij-x_||

                            float q = h * h - r * r;//(h^2-||rij||^2)
                            x_fric += m / ddens[sj] * MU * delxn * params.aw * q * q * q;//aw*q^3
                        }
                    }
                }
            }
        }
    }

    //i�̈ړ��ʂɂ�����x_friction�����̐��������߂�
    float3 norm_x_fric = normalize(x_fric);
    float3 dir_i_fric = norm_x_fric * dot(v_i, norm_x_fric);

    //printf("id %d friction delta x:%f,y:%f,z:%f\n",id, x_fric.x, x_fric.y, x_fric.z);

    float3 delta_x;
    if (length(dir_i_fric) <= length(x_fric)) {//�Î~���C�͂Ƃ��Ĉ����p�^�[��
        delta_x = -dir_i_fric;
    }
    else {//�����C�͂Ƃ��Ĉ����p�^�[��
        //delta_x = -dir_i_fric * min(length(x_fric) / length(dir_i_fric), 1.f);//[Macklin 2014]���Q�l�ɓK���ɋ��߂�
        delta_x = -x_fric * min(MU / length(x_fric), 1.0f);
        //delta_x = make_float3(0.f);
    }

    //�ʒu�C���ɂ��X�V
    dpos[DIM * sid] += delta_x.x;
    dpos[DIM * sid + 1] += delta_x.y;
    dpos[DIM * sid + 2] += delta_x.z;

    //�L�сE����f����̃�x=����/l0���C�������p���ɊҌ�����D��̃G�b�W�Ɖ��̃G�b�W�̗����ɊҌ�
    //���̂��߂ɁC�{��sid����Ƌ����ɕ�����K�v���邪�C�ǂݏo�����珑�����݂܂łɏ������������߁C���񉻂��Ă����v�ł��낤�Ɛ����D

    //�L�сE����f����ł̃���
    float3 lambda = delta_x * len;

    float4 delta_quat1 = 2.f * quatProduct(make_float4(lambda, 0.f), quatProduct(quat1, make_float4(0.f, 0.f, -1.f, 0.f)));
    float4 delta_quat2 = -2.f * quatProduct(make_float4(lambda, 0.f), quatProduct(quat2, make_float4(0.f, 0.f, -1.f, 0.f)));

    float4 new_quat1 = normalize(quat1 + delta_quat1);
    float4 new_quat2 = normalize(quat2 + delta_quat2);

    if (dfix[sid - 1] == 1) {
        dquat[QUAT * (sid - 1)] = new_quat1.x;
        dquat[QUAT * (sid - 1) + 1] = new_quat1.y;
        dquat[QUAT * (sid - 1) + 2] = new_quat1.z;
        dquat[QUAT * (sid - 1) + 3] = new_quat1.w;
    }
    if (dfix[sid + 1] == 1) {
        dquat[QUAT * sid] = new_quat2.x;
        dquat[QUAT * sid + 1] = new_quat2.y;
        dquat[QUAT * sid + 2] = new_quat2.z;
        dquat[QUAT * sid + 3] = new_quat2.w;
    }
}

//�X�̗��q�Ƃ̖��C���l���C�X�ɐÎ~���C���l����
//���C����͂Ƃ肠�����C���x�X�V�̍ۂɁC��x�̂ݍs���悤�ɐݒ肷��D
//��������ꍇ�ɂ�XPBD�ɂ��Ȃ��ƁC�����Ɉˑ����ďC���ʂ��ω�����ƍl�����邪�C���C����͐������C�����킯�ł͂Ȃ��D
//���ӗ��q���厖�Ȃ̂ŁCSort���ꂽ�ʒu�𗘗p����
//dpos:�ʒu
//dcurpos:�ʒu�C���O�̈ʒu
//drestdens:����x
//dvol:�̐�(���ʂ����z���ʂ����_0*V�Œ�`)
//ddens:���݂̖��x
//dfix:dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��
__global__
void CxFrictionAllParticlesConstraint(float* dpos, float* dcurpos, float* drestdens, float* dvol, float* ddens, int* dfix, int n) {
    // �O���b�h,�u���b�N���̃X���b�h�ʒu�𗱎q�C���f�b�N�X�Ƃ���
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= n) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�)

    float3 pos_i = params.cell.dSortedPos[id];
    float h = params.effective_radius;
    //�C���f�b�N�X�̌v�Z
    uint sid = params.cell.dSortedIndex[id];

    if (dfix[sid] == 1) return;//�Œ�_�ł���΁C�X�L�b�v

    //�O�X�e�b�v�̈ʒu
    float3 cur_pos_i = make_float3(dcurpos[sid * DIM], dcurpos[sid * DIM + 1], dcurpos[sid * DIM + 2]);
    //�ʒu�C���Ȃǂɂ��ړ���
    float3 v_i = pos_i - cur_pos_i;
    //�Î~���C�W��
    float mu = MU;

    // ���q�𒆐S�Ƃ��Ĕ��ah���Ɋ܂܂��O���b�h(caclGridPos���ŋ��E��������)
    int3 grid_pos0, grid_pos1;
    grid_pos0 = calcGridPos(pos_i - make_float3(h));
    grid_pos1 = calcGridPos(pos_i + make_float3(h));

    //�ŏI�I�Ȗ��C��
    float3 x_fric = make_float3(0.f);

    for (int z = grid_pos0.z; z <= grid_pos1.z; ++z) {
        for (int y = grid_pos0.y; y <= grid_pos1.y; ++y) {
            for (int x = grid_pos0.x; x <= grid_pos1.x; ++x) {
                int3 ngrid = make_int3(x, y, z);
                uint ghash = calcGridHash(ngrid);   // �O���b�h�n�b�V���l
                // �Z�����̃p�[�e�B�N���̃X�^�[�g�C���f�b�N�X
                uint startIndex = params.cell.dCellStart[ghash];
                if (startIndex != 0xffffffff) {	// �Z������łȂ����̃`�F�b�N
                    // �Z�����̃p�[�e�B�N���Ŕ���
                    uint endIndex = params.cell.dCellEnd[ghash];
                    for (uint j = startIndex; j < endIndex; ++j) {
                        uint sj = params.cell.dSortedIndex[j];
                        float3 pos_j = params.cell.dSortedPos[j];

                        //�O�X�e�b�v�̈ʒu
                        float3 cur_pos_j = make_float3(dcurpos[sj * DIM], dcurpos[sj * DIM + 1], dcurpos[sj * DIM + 2]);
                        //j�̗��q�̏������x
                        float restdens_j = drestdens[sj];
                        //j�̑̐�
                        float vol_j = dvol[sj];
                        //����
                        float m = restdens_j * vol_j;

                        float3 r_ij = pos_i - pos_j;
                        float r = length(r_ij);
                        if (r <= 1.0e-3) continue;
                        if (r < h) {
                            //�����ɖ��C���������
                            float3 v_j = pos_j - cur_pos_j;

                            float3 v_ij = v_i - v_j;

                            r_ij = normalize(r_ij);

                            //�Փ˖@���ɑ΂��Đ����Ȑ��������߂�
                            float3 delxn = v_ij - r_ij * dot(v_ij, r_ij);//delta x_��=v_ij-x_||

                            float q = h * h - r * r;//(h^2-||rij||^2)
                            float3 tmp_x_fric = m / ddens[sj] * MU * delxn * params.aw * q * q * q;//aw*q^3

                            x_fric -= tmp_x_fric;

                            //�Î~���C�̎c�蕨
                            ////���C�͂𐳋K�����ĕ����x�N�g����
                            //float3 norm_tmp_x_fric = normalize(tmp_x_fric);
                            ////v_i�̂����C���C�͂̕����̐��������o��
                            //float3 dir_i_fric = norm_tmp_x_fric * dot(v_i, norm_tmp_x_fric);

                            ////�Î~���C�͂Ȃ炻����̕����̐�����ł�����
                            //if (length(dir_i_fric) <= length(tmp_x_fric)) {
                            //    x_fric -= dir_i_fric;
                            //}
                            ////�����C�Ȃ�C���̂܂ܓK�p���邱�ƂƂ���
                            //else {
                            //    x_fric -= tmp_x_fric;
                            //}
                        }
                    }
                }
            }
        }
    }

    //�ʒu�C���ɂ��X�V
    dpos[DIM * sid] += x_fric.x;
    dpos[DIM * sid + 1] += x_fric.y;
    dpos[DIM * sid + 2] += x_fric.z;
}

//�V����2���_����Ԃ̃G�b�W�̎p�������߂�
//dpos:�ʒu
//dquat:�p��
//dfix:�Œ�_(�є��̊J�n�_)�������z��(1�Ȃ�Œ�_,0�Ȃ炻��ȊO)
//n:���q��
__global__
void CxQuatSet(float* dpos, float* dquat, int* dfix, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n - 1) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    if (dfix[id] == 1 || dfix[id + 1] == 1) return;//�e���̖��ɕӂ̐������s��

    float3 pos0 = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 pos1 = make_float3(dpos[DIM * (id + 1)], dpos[DIM * (id + 1) + 1], dpos[DIM * (id + 1) + 2]);
    //2���_�̊Ԃ̕����x�N�g��
    float3 dir = pos1 - pos0;
    //�G�b�W�̎p�������߂�
    float4 quat = quatFromDirector(dir);

    dquat[QUAT * id] = quat.x;
    dquat[QUAT * id + 1] = quat.y;
    dquat[QUAT * id + 2] = quat.z;
    dquat[QUAT * id + 3] = quat.w;
}

//���͂ɂ��g���N�̌v�Z������
__global__
void CxCalcTorque(float* dpos,float* dmas,float* dquat, float* dfss,float* dlength,float* dkss, int* dfix,float3 gravity, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= n - 1) return; // ���q���𒴂���X���b�hID�̃`�F�b�N(�]�肪�o�Ȃ��悤�Ƀu���b�N���Ȃǂ��ݒ�ł���Ȃ�K�v�Ȃ�) �Ō�̗��q�̓X�L�b�v
    if (dfix[id] == 1 || dfix[id + 1] == 1) return;//�ŏ��̓�̗��q�͌Œ�_�Ƃ��Ĉ���

    float3 pos0 = make_float3(dpos[DIM * (id - 1)], dpos[DIM * (id - 1) + 1], dpos[DIM * (id - 1) + 2]);
    float3 pos1 = make_float3(dpos[DIM * id], dpos[DIM * id + 1], dpos[DIM * id + 2]);
    float3 mid = (pos0 + pos1) / 2.f;

    float4 quat = make_float4(dquat[QUAT * id], dquat[QUAT * id + 1], dquat[QUAT * id + 2], dquat[QUAT * id + 3]);
    float3 fss = make_float3(dfss[DIM * id], dfss[DIM * id + 1], dfss[DIM * id + 2]);
    //�p����e3����0->1�̕����x�N�g�������߁C����͒P�ʃx�N�g���Ȃ̂ŁC���������߂�K�v������D
    float len = dlength[id];

    float mass0 = dmas[id-1];
    float mass1 = dmas[id];

    float3 torque_calc = CalcNormalTorque(pos0, pos1, quat, fss, len, mass0, mass1, make_float3(0.f, -9.81, 0.f));
    printf("id %d calcTorque x:%f,y:%f,z:%f\n", id, torque_calc.x, torque_calc.y, torque_calc.z);

    float4 torque = StretchingShearTorque(quat, pos0, pos1, len, dkss[id]);
    //printf("id %d StretchingShearTorque x:%f,y:%f,z:%f,w:%f\n",id, torque.x, torque.y, torque.z, torque.w);
}
